#include "hip/hip_runtime.h"
/* -*- mode: C++; c-file-style: "bsd"; c-basic-offset: 2; indent-tabs-mode: nil -*- */

#include "linearSolvers.hpp"

#include <stdio.h>

#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipblas.h>
#include <hipsolver.h>

#include "Complex.hpp"
#include "Matrix.hpp"
#include "Accelerator/DeviceStorage.hpp"
#include "Accelerator/deviceCheckError.hpp"


/*
#define IDX(i, j, lDim) (((j)*(lDim))+(i))

template <typename T>
void zeroMatrixCuda(T *devM, int lDim, int nCol)
{
//  for(int i=0; i<m.n_row(); i++)
//    for(int j=0; j<m.n_col(); j++)
//      m(i,j) = 0.0;
  hipMemset(devM, 0, lDim*nCol*sizeof(T));
}

template <typename T>
__global__ void setDiagonalKernelCuda(T *devM, int lDim, int nCol, T val)
{
  int i=blockIdx.x*blockDim.x + threadIdx.x;
  if(i<nCol)
  {
    devM[IDX(i, i, lDim)] = val;
  }
}

template <typename T>
__global__ void addDiagonalKernelCuda(T *devM, int lDim, int nCol, T val)
{
  int i=blockIdx.x*blockDim.x + threadIdx.x;
  if(i<nCol)
  {
    devM[IDX(i, i, lDim)] = hipCadd(devM[IDX(i, i, lDim)], val);
  }
}

template <typename T>
void unitMatrixCuda(T *devM, int lDim, int nCol)
{
  zeroMatrixCuda(devM, lDim, nCol);
  setDiagonalKernelCuda<<<nCol,1>>>(devM, lDim, nCol, 1.0);
}

*/



template <typename T>
__global__ void zeroDiagonalBlocksKernelCuda(T *devM, int lDim, int nCol, int blockSize)
{
  int iBlock = blockIdx.x*blockDim.x + threadIdx.x;
  int jBlock = blockIdx.y*blockDim.y + threadIdx.y;
  if(iBlock<nCol/blockSize)
    if(jBlock<nCol/blockSize)
    {
      int ii=iBlock*blockSize;
      int jj=jBlock*blockSize;
      for(int i=0; i<std::min(blockSize, nCol-ii); i++)
        for(int j=0; j<std::min(blockSize, nCol-jj); j++)
          devM[IDX(ii+i, jj+j, lDim)] = 0.0;
    }
}

void transferT0MatrixToGPUCuda(Complex *devT0, LSMSSystemParameters &lsms, LocalTypeInfo &local,
                               AtomData &atom, int iie, int ispin)
{
  int kkrsz_ns = lsms.n_spin_cant*atom.kkrsz;

  int jsm =  kkrsz_ns * kkrsz_ns * ispin;

  hipMemcpy(devT0, &local.tmatStore(iie*local.blkSizeTmatStore + jsm,atom.LIZStoreIdx[0]),
             kkrsz_ns*kkrsz_ns*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);

#ifdef T_CUDA_DEBUG
  std::ofstream myfile;
  auto filename = "TAU" + std::to_string(iie) + "_" + std::to_string(ispin);
  myfile.open(filename.c_str());
  for (int i = 0; i < kkrsz_ns * kkrsz_ns; i++) {
    Complex buffer;
    hipMemcpy(&buffer, &devT0[i], sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    myfile << std::real(buffer) << " " << std::imag(buffer) << std::endl;
  }
  myfile.close();
#endif

}

void transferMatrixToGPUCuda(Complex *devM, Matrix<Complex> &m)
{
  hipMemcpy(devM, &m(0,0), m.l_dim()*m.n_col()*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
}

void transferMatrixFromGPUCuda(Matrix<Complex> &m, hipDoubleComplex *devM)
{
  hipMemcpy(&m(0,0), devM,  m.l_dim()*m.n_col()*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
}

__global__ void copyTMatrixToTauCuda(hipDoubleComplex *tau, hipDoubleComplex *t, int kkrsz, int nrmat)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i < kkrsz)
  {
    for(int j=0; j<kkrsz; j++)
      tau[IDX(i,j,nrmat)] = t[IDX(i,j,kkrsz)];
  }
}


__global__ void copyTauToTau00Cuda(hipDoubleComplex *tau00, hipDoubleComplex *tau, int kkrsz, int nrmat)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if(i < kkrsz)
  {
    for(int j=0; j<kkrsz; j++)
      tau00[IDX(i,j,kkrsz)] = tau[IDX(i,j,nrmat)];
  }
}

void solveTau00zgetrf_cublas(LSMSSystemParameters &lsms, LocalTypeInfo &local, DeviceStorage &d, AtomData &atom,
                             Complex *tMatrix, Complex *devM,
                             Matrix<Complex> &tau00)
{
  hipblasHandle_t cublasHandle = DeviceStorage::getCublasHandle();
  int nrmat_ns = lsms.n_spin_cant*atom.nrmat; // total size of the kkr matrix
  int kkrsz_ns = lsms.n_spin_cant*atom.kkrsz; // size of t00 block
  // reference algorithm. Use LU factorization and linear solve for dense matrices in LAPACK
  hipDoubleComplex *Aarray[1], *Barray[1];

  hipDoubleComplex *devTau = (hipDoubleComplex *)d.getDevTau();
  hipDoubleComplex *devTau00 = (hipDoubleComplex *)d.getDevTau00();
  // printf("zero Matrix\n");
  zeroMatrixCuda(devTau, nrmat_ns, kkrsz_ns);
  deviceCheckError();
  // printf("copyTMatrixToTau\n");
  copyTMatrixToTauCuda<<<kkrsz_ns,1>>>(devTau, (hipDoubleComplex *)tMatrix, kkrsz_ns, nrmat_ns);
  deviceCheckError();

  Barray[0] = devTau;

  Aarray[0] = (hipDoubleComplex *)devM;

  int *ipivArray=d.getDevIpvt();
  int *infoArray = d.getDevInfo();
  int info;

  // printf("hipblasZgetrfBatched\n");
  cublasCheckError(hipblasZgetrfBatched(cublasHandle, nrmat_ns, Aarray, nrmat_ns, ipivArray, infoArray, 1));
  // printf("hipblasZgetrsBatched\n");

  cublasCheckError(hipblasZgetrsBatched(cublasHandle, HIPBLAS_OP_N, nrmat_ns, kkrsz_ns, Aarray, nrmat_ns, ipivArray,
                                 Barray, nrmat_ns, &info, 1));

  // copy result into tau00
  // printf("copyTauToTau00\n");
  copyTauToTau00Cuda<<<kkrsz_ns,1>>>(devTau00, devTau, kkrsz_ns, nrmat_ns);
  deviceCheckError();
  // printf("transferMatrixFromGPU\n");
  transferMatrixFromGPUCuda(tau00, devTau00);
  deviceCheckError();
}

#ifndef ARCH_IBM
void solveTau00zzgesv_cusolver(LSMSSystemParameters &lsms, LocalTypeInfo &local, DeviceStorage &d, AtomData &atom,
                               Complex *tMatrix, Complex *devM, Matrix<Complex> &tau00, int ispin)
{
  hipsolverHandle_t cusolverDnHandle = DeviceStorage::getCusolverDnHandle();
  int nrmat_ns = lsms.n_spin_cant*atom.nrmat; // total size of the kkr matrix
  int kkrsz_ns = lsms.n_spin_cant*atom.kkrsz; // size of t00 block
  // reference algorithm. Use LU factorization and linear solve for dense matrices in LAPACK

  hipDoubleComplex *devTau = (hipDoubleComplex *)d.getDevTau();
  hipDoubleComplex *devTau00 = (hipDoubleComplex *)d.getDevTau00();
  hipDoubleComplex *devWork = (hipDoubleComplex *)d.getDevWork();

  hipDoubleComplex *devT = (hipDoubleComplex *)d.getDevT();

  int *devIpiv = d.getDevIpvt();
  int devInfo[1]; // d.getDevInfo();

  zeroMatrixCuda(devTau, nrmat_ns, kkrsz_ns);
  zeroMatrixCuda(devT, nrmat_ns, kkrsz_ns);
  copyTMatrixToTauCuda<<<kkrsz_ns,1>>>(devT, (hipDoubleComplex *)tMatrix, kkrsz_ns, nrmat_ns);

  int iter;

  hipsolverStatus_t status = hipsolverDnZZgesv(cusolverDnHandle, nrmat_ns, kkrsz_ns,
                                             (hipDoubleComplex *)devM, nrmat_ns, devIpiv, devT, nrmat_ns, devTau, nrmat_ns,
                                             devWork, d.getDevWorkBytes(), &iter, devInfo);

  if(status!=HIPSOLVER_STATUS_SUCCESS)
  {
    printf("hipsolverDnZZgesv returned %d\n",status);
  }

  copyTauToTau00Cuda<<<kkrsz_ns,1>>>(devTau00, devTau, kkrsz_ns, nrmat_ns);
  transferMatrixFromGPUCuda(tau00, devTau00);
}
#endif

void solveTau00zgetrf_cusolver(LSMSSystemParameters &lsms, LocalTypeInfo &local, DeviceStorage &d, AtomData &atom,
                               Complex *tMatrix, Complex *devM, Matrix<Complex> &tau00, int ispin)
{
  hipsolverHandle_t cusolverDnHandle = DeviceStorage::getCusolverDnHandle();
  int nrmat_ns = lsms.n_spin_cant*atom.nrmat; // total size of the kkr matrix
  int kkrsz_ns = lsms.n_spin_cant*atom.kkrsz; // size of t00 block
  // reference algorithm. Use LU factorization and linear solve for dense matrices in LAPACK
  hipDoubleComplex *devTau = (hipDoubleComplex *)d.getDevTau();
  hipDoubleComplex *devTau00 = (hipDoubleComplex *)d.getDevTau00();
  hipDoubleComplex *devWork = (hipDoubleComplex *)d.getDevWork();

  int *devIpiv = d.getDevIpvt();
  int *devInfo = d.getDevInfo();

  zeroMatrixCuda(devTau, nrmat_ns, kkrsz_ns);
  deviceCheckError();
  copyTMatrixToTauCuda<<<kkrsz_ns,1>>>(devTau, (hipDoubleComplex *)tMatrix, kkrsz_ns, nrmat_ns);
  deviceCheckError();

  cusolverCheckError(hipsolverDnZgetrf(cusolverDnHandle, nrmat_ns, nrmat_ns,
                                      (hipDoubleComplex *)devM, nrmat_ns, devWork, devIpiv,
                                      devInfo ));


  cusolverCheckError(hipsolverDnZgetrs(cusolverDnHandle, HIPBLAS_OP_N, nrmat_ns, kkrsz_ns,
                                      (hipDoubleComplex *)devM, nrmat_ns, devIpiv, devTau, nrmat_ns, devInfo));

  // copy result into tau00
  copyTauToTau00Cuda<<<kkrsz_ns,1>>>(devTau00, devTau, kkrsz_ns, nrmat_ns);
  deviceCheckError();
  transferMatrixFromGPUCuda(tau00, devTau00);
  deviceCheckError();
}

#ifdef USE_XGETRF
void solveTau00Xgetrf_cusolver(LSMSSystemParameters &lsms, LocalTypeInfo &local, DeviceStorage &d, AtomData &atom,
                               Complex *tMatrix, Complex *devM, Matrix<Complex> &tau00, int ispin)
{
  hipsolverHandle_t cusolverDnHandle = DeviceStorage::getCusolverDnHandle();
  hipsolverDnParams_t cusolverDnParams = DeviceStorage::getCusolverParams();
  int nrmat_ns = lsms.n_spin_cant*atom.nrmat; // total size of the kkr matrix
  int kkrsz_ns = lsms.n_spin_cant*atom.kkrsz; // size of t00 block
  // reference algorithm. Use LU factorization and linear solve for dense matrices in LAPACK
  hipDoubleComplex *devTau = (hipDoubleComplex *)d.getDevTau();
  hipDoubleComplex *devTau00 = (hipDoubleComplex *)d.getDevTau00();
  void *devWork = d.getDevWork();
  size_t devWorkBytes = d.getDevWorkBytes();
  int64_t *devIpiv=d.getDevIpvt64();
  void *hostWork = d.getHostWork();
  size_t hostWorkBytes = d.getHostWorkBytes();
  int *devInfo = d.getDevInfo();

  zeroMatrixCuda(devTau, nrmat_ns, kkrsz_ns);
  deviceCheckError();
  copyTMatrixToTauCuda<<<kkrsz_ns,1>>>(devTau, (hipDoubleComplex *)tMatrix, kkrsz_ns, nrmat_ns);
  deviceCheckError();

  cusolverCheckError(hipsolverDnXgetrf(cusolverDnHandle,
                                      cusolverDnParams,
                                      (int64_t)nrmat_ns,
                                      (int64_t)nrmat_ns,
                                      HIP_C_64F,
                                      (hipDoubleComplex *)devM,
                                      (int64_t)nrmat_ns,
                                      devIpiv,
                                      HIP_C_64F,
                                      devWork,
                                      devWorkBytes,
                                      hostWork,
                                      hostWorkBytes,
                                      devInfo));

  cusolverCheckError(hipsolverDnXgetrs(cusolverDnHandle,
                                      cusolverDnParams,
                                      HIPBLAS_OP_N,
                                      (int64_t)nrmat_ns,
                                      (int64_t)kkrsz_ns,
                                      HIP_C_64F,
                                      (hipDoubleComplex *)devM,
                                      (int64_t)nrmat_ns,
                                      devIpiv,
                                      HIP_C_64F,
                                      devTau,
                                      (int64_t)nrmat_ns,
                                      devInfo));
  
  // copy result into tau00
  copyTauToTau00Cuda<<<kkrsz_ns,1>>>(devTau00, devTau, kkrsz_ns, nrmat_ns);
  deviceCheckError();
  transferMatrixFromGPUCuda(tau00, devTau00);
  deviceCheckError();
}
#endif

#ifdef USE_IRSXGESV
void solveTau00IRSXgesv_cusolver(LSMSSystemParameters &lsms, LocalTypeInfo &local, DeviceStorage &d, AtomData &atom,
                                 Complex *tMatrix, Complex *devM, Matrix<Complex> &tau00, int ispin)
{
  hipsolverHandle_t cusolverDnHandle = DeviceStorage::getCusolverDnHandle();
  cusolverDnIRSParams_t cusolverDnIRSParams = DeviceStorage::getCusolverIRSParams();
  cusolverDnIRSInfos_t cusolverDnIRSInfo = DeviceStorage::getCusolverIRSInfo();
  int nrmat_ns = lsms.n_spin_cant*atom.nrmat; // total size of the kkr matrix
  int kkrsz_ns = lsms.n_spin_cant*atom.kkrsz; // size of t00 block
  // reference algorithm. Use LU factorization and linear solve for dense matrices in LAPACK
  hipDoubleComplex *devTau = (hipDoubleComplex *)d.getDevTau();
  hipDoubleComplex *devTau00 = (hipDoubleComplex *)d.getDevTau00();
  hipDoubleComplex *devWork = (hipDoubleComplex *)d.getDevWork();
  hipDoubleComplex *devX = (hipDoubleComplex *)d.getDevX();
  size_t devWorkBytes = d.getDevWorkBytes();
  int *devInfo = d.getDevInfo();

  zeroMatrixCuda(devTau, nrmat_ns, kkrsz_ns);
  deviceCheckError();
  copyTMatrixToTauCuda<<<kkrsz_ns,1>>>(devTau, (hipDoubleComplex *)tMatrix, kkrsz_ns, nrmat_ns);
  deviceCheckError();

  cusolverDnIRSInfos_t info;
  int niters;
  cusolverCheckError(cusolverDnIRSXgesv(cusolverDnHandle,
                                        cusolverDnIRSParams,
                                        cusolverDnIRSInfo,
                                        nrmat_ns,
                                        kkrsz_ns,
                                        (hipDoubleComplex *)devM,
                                        nrmat_ns,
                                        devTau,
                                        nrmat_ns,
                                        devX,
                                        nrmat_ns,
                                        devWork,
                                        devWorkBytes,
                                        &niters,
                                        devInfo));

  // copy result into tau00
  copyTauToTau00Cuda<<<kkrsz_ns,1>>>(devTau00, devX, kkrsz_ns, nrmat_ns);
  deviceCheckError();
  transferMatrixFromGPUCuda(tau00, devTau00);
  deviceCheckError();
}
#endif
