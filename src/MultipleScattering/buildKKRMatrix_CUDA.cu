#include "hip/hip_runtime.h"
/* -*- mode: C++; c-file-style: "bsd"; c-basic-offset: 2; indent-tabs-mode: nil -*- */

#include "buildKKRMatrix.hpp"

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <string>

#include "Complex.hpp"
#include "Matrix.hpp"
#include <vector>

#include "Accelerator/DeviceStorage.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include "cuComplexOperators.hpp"
#include <hipblas.h>
#include <hipsolver.h>

#include "linearSolvers.hpp"

// we might want to distinguish between systems where all lmax (and consequently kkrsz_ns) are the same
// and systems with potential different lmax on different atoms and l steps

// #define COMPARE_ORIGINAL 1

// Fortran layout for matrix
// #define IDX(i, j, lDim) (((j)*(lDim))+(i))
#define IDX3(i, j, k, lDim, mDim) (((k)*(lDim)*(mDim)) + ((j)*(lDim)) + (i))

__device__
__inline__
void calculateHankelCuda(hipDoubleComplex prel, double r, int lend, hipDoubleComplex *ilp1, hipDoubleComplex *hfn)
{
  if(threadIdx.x == 0)
  {
    const hipDoubleComplex sqrtm1 = make_hipDoubleComplex(0.0, 1.0);
    hipDoubleComplex z = prel * make_hipDoubleComplex(r,0.0);
    hfn[0] = make_hipDoubleComplex(0.0, -1.0); //-sqrtm1;
    hfn[1] = -1.0 - sqrtm1/z;
    for(int l=1; l<lend; l++)
    {
      hfn[l+1] = ((2.0*l+1.0) * hfn[l]/z) - hfn[l-1];
    }

//             l+1
//     hfn = -i   *h (k*R  )*sqrt(E)
//                  l    ij

    z = exp(sqrtm1*z)/r;
    for(int l=0; l<=lend; l++)
    {
      hfn[l] = ((-hfn[l]) * z) * ilp1[l];
    }
  }
//  __syncthreads();
}

__device__
__inline__
void calculateSinCosPowersCuda(Real *rij, int lend, Real *sinmp, Real *cosmp)
{
  const Real ptol = 1.0e-6;
  Real pmag = std::sqrt(rij[0]*rij[0]+rij[1]*rij[1]);
  cosmp[0] = 1.0;
  sinmp[0] = 0.0;
  if(pmag>ptol)
  {
    cosmp[1] = rij[0]/pmag;
    sinmp[1] = rij[1]/pmag;
  } else {
    cosmp[1] = 0.0;
    sinmp[1] = 0.0;
  }
  for(int m=2; m<=lend; m++)
  {
    cosmp[m] = cosmp[m-1]*cosmp[1] - sinmp[m-1]*sinmp[1];
    sinmp[m] = sinmp[m-1]*cosmp[1] + cosmp[m-1]*sinmp[1];
  }
}

// __device__ __inline__ int plmIdxDev(int l, int m)
// { return l*(l+1)/2+m; }

#define PLM_IDX(l,m) (((l)*((l)+1))/2 + (m))

__device__
__inline__
void associatedLegendreFunctionNormalizedCuda(Real x, int lmax, Real *Plm)
{
  const Real pi = std::acos(-1.0);
  // y = \sqrt{1-x^2}
  Real y = std::sqrt(1.0-x*x);
  // initialize the first entry
  // Plm[0]=std::sqrt(R(1)/(R(2)*pi));
  Plm[0]=std::sqrt(1.0/(4.0*pi));

  if(lmax<1) return;

  for(int m=1; m<=lmax; m++)
  {
    // \bar{P}_{mm} = - \sqrt{\frac{2m+1}{2m}} y \bar{P}_{m-1, m-1}
    Plm[PLM_IDX(m,m)] = - std::sqrt(Real(2*m+1)/Real(2*m)) * y * Plm[PLM_IDX(m-1,m-1)];
    // \bar{P}_{mm-1} = \sqrt{2 m + 1} x \bar{P}_{m-1, m-1}
    Plm[PLM_IDX(m,m-1)] = std::sqrt(Real(2*m+1)) * x * Plm[PLM_IDX(m-1,m-1)];
  }

  for(int m=0; m<lmax; m++)
  {
    for(int l=m+2; l<=lmax; l++)
    {
      // \bar{P}_{lm} = a_{lm} (x \bar{P}_{l-1. m} - b_{lm} \bar{P}_{l-2, m})
      // a_{lm} = \sqrt{\frac{(4 l^2 - 1)(l^2 - m^2)}}
      // b_{lm} = \sqrt{\frac{(l -1)^2 - m^2}{4 (l-1)^2 -1}}
      Real a_lm = std::sqrt(Real(4*l*l-1)/Real(l*l - m*m));
      Real b_lm = std::sqrt(Real((l-1)*(l-1) - m*m)/Real(4*(l-1)*(l-1)-1));
      Plm[PLM_IDX(l,m)] = a_lm * (x * Plm[PLM_IDX(l-1,m)] - b_lm * Plm[PLM_IDX(l-2,m)]);
    }
  }
}

__device__
__inline__
hipDoubleComplex dlmFunction(hipDoubleComplex *hfn, double *cosmp, double *sinmp, double *plm, int l, int m)
{
  int mAbs = abs(m);

  hipDoubleComplex dlm = hfn[l]*plm[PLM_IDX(l,mAbs)];
  if(m==0) return dlm;

  if(m<0)
  {
    dlm = dlm * make_hipDoubleComplex(cosmp[mAbs],sinmp[mAbs]);
    if((mAbs & 0x01) != 0) // m is odd
      dlm = -dlm;
  } else {
    dlm = dlm * make_hipDoubleComplex(cosmp[mAbs],-sinmp[mAbs]);
  }

  return dlm;
}


size_t sharedMemoryBGijCuda(LSMSSystemParameters &lsms, size_t *hfnOffset, size_t *sinmpOffset, size_t *cosmpOffset,
                            size_t *plmOffset, size_t *dlmOffset)
{
  size_t size = 0;

  *hfnOffset = size;
  size += sizeof(hipDoubleComplex) * (2*lsms.maxlmax + 1);

  *sinmpOffset = size;
  size += sizeof(double) * (2*lsms.maxlmax + 1);

  *cosmpOffset = size;
  size += sizeof(double) * (2*lsms.maxlmax + 1);

  *plmOffset = size;
  size += sizeof(double) * (lsms.angularMomentumIndices.ndlm);

//  *dlmOffset = size;
//  size += sizeof(hipDoubleComplex) * (lsms.angularMomentumIndices.ndlj);

  return size;
}


__global__
void setBGijCuda(bool fullRelativity, int n_spin_cant, int *LIZlmax,
                 int *offsets, size_t nrmat_ns, hipDoubleComplex *devBgij)
{
  if(n_spin_cant == 1) return;

  int ir1 = blockIdx.x;
  int ir2 = blockIdx.y;
  int iOffset = offsets[ir1];
  int jOffset = offsets[ir2];

  int kkri=(LIZlmax[ir1]+1)*(LIZlmax[ir1]+1);
  int kkrj=(LIZlmax[ir2]+1)*(LIZlmax[ir2]+1);

  if(!fullRelativity) //(lsms.relativity != full)
  {
    for(int ij=threadIdx.x; ij < kkri*kkrj; ij += blockDim.x)
    {
      int i = ij % kkri;
      int j = ij / kkri;
/*
    for(int i=0; i<kkri; i++)
      for(int j=0; j<kkrj; j++)
      {
*/
      devBgij[IDX(iOffset + kkri + i, jOffset        + j, nrmat_ns)] = make_hipDoubleComplex(0.0, 0.0); // bgij(iOffset + i, jOffset + j);
      devBgij[IDX(iOffset        + i, jOffset + kkrj + j, nrmat_ns)] = make_hipDoubleComplex(0.0, 0.0); // bgij(iOffset + i, jOqffset + j);
      devBgij[IDX(iOffset + kkri + i, jOffset + kkrj + j, nrmat_ns)] = devBgij[IDX(iOffset + i, jOffset + j, nrmat_ns)];
    }
  } else {
    /*
            call relmtrx(gij,bgij,kkr1,kkr2)
            fac=psq/ce
            do i=1,kkr1_ns
              do j=1,kkr2_ns
                bgij(i,j)=fac*bgij(i,j)
              end do
            end do
    */
    printf("Fully relativistic calculation not yet implemented in 'MultipleScattering/buildKKRMatrix.cpp : setBGijCPU'\n");
    // exit(1);
  }
}


__global__
void buildGijCudaKernel(Real *LIZPos, int *LIZlmax, int *lofk, int *mofk, hipDoubleComplex *ilp1, hipDoubleComplex *illp, Real *cgnt,
                        int ndlj_illp, int lmaxp1_cgnt, int ndlj_cgnt,
                         size_t hfnOffset, size_t sinmpOffset, size_t cosmpOffset, size_t plmOffset, size_t dlmOffset,
#if !defined(COMPARE_ORIGINAL)
                         hipDoubleComplex energy, hipDoubleComplex prel, int *offsets, size_t nrmat_ns, hipDoubleComplex *devBgij)
#else
                          hipDoubleComplex energy, hipDoubleComplex prel, int *offsets, size_t nrmat_ns, hipDoubleComplex *devBgij, char *testSM)
#endif
//  void buildBGijCPU(LSMSSystemParameters &lsms, AtomData &atom, int ir1, int ir2, Real *rij,
//                  Complex energy, Complex prel, int iOffset, int jOffset, Matrix<Complex> &bgij)
{
  int ir1 = blockIdx.x;
  int ir2 = blockIdx.y;
  extern char __shared__ sharedMemory[];

  if(ir1 != ir2)
  {
    int iOffset = offsets[ir1];
    // int iOffset = ir1 * kkrsz_ns;
    int jOffset = offsets[ir2];
    // int jOffset = ir2 * kkrsz_ns;

    Real rij[3];
    rij[0] = LIZPos[3*ir1 + 0] - LIZPos[3*ir2 + 0];
    rij[1] = LIZPos[3*ir1 + 1] - LIZPos[3*ir2 + 1];
    rij[2] = LIZPos[3*ir1 + 2] - LIZPos[3*ir2 + 2];

    // Complex hfn[2*lsms.maxlmax + 1];
    hipDoubleComplex *hfn = (hipDoubleComplex *) (sharedMemory + hfnOffset);
    // Real sinmp[2*lsms.maxlmax + 1];
    Real *sinmp = (Real *) (sharedMemory + sinmpOffset);
    // Real cosmp[2*lsms.maxlmax + 1];
    Real *cosmp = (Real *) (sharedMemory + cosmpOffset);
    // Real plm[lsms.angularMomentumIndices.ndlm];
    Real *plm = (Real *) (sharedMemory + plmOffset);
    // Complex dlm[lsms.angularMomentumIndices.ndlj];
    // hipDoubleComplex *dlm = (hipDoubleComplex *) (sharedMemory + dlmOffset);

#if defined(COMPARE_ORIGINAL)
    hipDoubleComplex *testHfn = (hipDoubleComplex *) (testSM + hfnOffset);
    Real *testSinmp = (Real *) (testSM + sinmpOffset);
    Real *testCosmp = (Real *) (testSM + cosmpOffset);
    Real *testPlm = (Real *) (testSM + plmOffset);
    hipDoubleComplex *testDlm = (hipDoubleComplex *) (testSM + dlmOffset);
#endif

    Real r = std::sqrt(rij[0]*rij[0] + rij[1]*rij[1] + rij[2]*rij[2]);
    int lmax1 = LIZlmax[ir1];
    int lmax2 = LIZlmax[ir2];
    int kkri=(lmax1+1)*(lmax1+1);
    int kkrj=(lmax2+1)*(lmax2+1);
    int lend = lmax1 + lmax2;

    Real pi4=4.0*2.0*std::asin(1.0);
    Real cosTheta = rij[2]/r;

    if(threadIdx.x == 0)
    {
      calculateHankelCuda(prel, r, lend, ilp1, hfn);

      associatedLegendreFunctionNormalizedCuda(cosTheta, lend, plm);

      // for associatedLegendreFunctionNormalized all clm[i] == 1.0
      // for(int j=0;j<ndlm_local;j++)
      //   plm[j]=clm[j]*plm[j];

      //     calculate cos(phi) and sin(phi) .................................
      // needs to be serial
      calculateSinCosPowersCuda(rij, lend, sinmp, cosmp);
    }
    __syncthreads();

/*
    // can be parallel
    int j;
    int ll;
//    for(int l = threadIdx.x; l<=lend; l += blockDim.x)
    if(threadIdx.x == 0)
    {
      for(int l = 0; l<=lend; l++)
      {
        // int ll = l*(l+1);
        // j = ll;
        // ll = ll/2;
        j = l*(l+1);
        ll = j/2;
        double m1m = 1.0;
        dlm[j] = hfn[l]*plm[ll];

        for(int m=1; m<=l; m++)
        {
          m1m = -m1m;
          hipDoubleComplex fac = plm[ll+m] * make_hipDoubleComplex(cosmp[m],sinmp[m]);
          dlm[j-m] = hfn[l]*m1m*fac;
          dlm[j+m] = hfn[l]*hipConj(fac);
        }

      }
    }
    __syncthreads();
*/

#if defined(COMPARE_ORIGINAL)
    if(ir1 == 0 && ir2 == 1 && threadIdx.x == 0)
    {
      for(int l = 0; l<=lend; l++)
      {
        testHfn[l] = hfn[l];
        testSinmp[l] = sinmp[l];
        testCosmp[l] = cosmp[l];
      }
    }
#endif

//     ================================================================
//     calculate g(R_ij)...............................................
  // for(int i=0; i<kkri*kkrj; i++) gij[i]=0.0;

    // for(int i=0; i<kkri; i++)
    //   for(int j=0; j<kkrj; j++)
    // for(int ij=0; ij < kkri*kkrj; ij++)
    for(int ij=threadIdx.x; ij < kkri*kkrj; ij += blockDim.x)
    {
      int lm2 = ij % kkri;
      int lm1 = ij / kkri;
      devBgij[IDX(iOffset + lm2, jOffset + lm1, nrmat_ns)] = make_hipDoubleComplex(0.0, 0.0);
      // bgij(iOffset + lm2, jOffset + lm1) = 0.0;
      // }

//     loop over l1,m1............................................
//    for(int lm1=0; lm1<kkrj; lm1++)
//    {
      int l1=lofk[lm1];
      int m1=mofk[lm1];

//        loop over l2,m2..............................................
//      for(int lm2=0; lm2<kkri; lm2++)
//      {
      int l2=lofk[lm2];
      int m2=mofk[lm2];

//          ==========================================================
//          l2-l1
//          illp(lm2,lm1) = i
//
//          perform sum over l3 with gaunt # ......................
//          ==========================================================

      int m3=m2-m1;
      int llow=max(abs(m3), abs(l1-l2));
      if(hipCabs(prel)==0.0) llow=l1+l2;
      for(int l3=l1+l2; l3>=llow; l3-=2)
      {
        int j=l3*(l3+1)+m3;
        // gij[lm2+lm1*kkri] = gij[lm2+lm1*kkri]+cgnt(l3/2,lm1,lm2)*dlm[j];
        devBgij[IDX(iOffset + lm2, jOffset + lm1, nrmat_ns)] =  devBgij[IDX(iOffset + lm2, jOffset + lm1, nrmat_ns)]
          + cgnt[IDX3(l3/2,lm1,lm2,lmaxp1_cgnt,ndlj_cgnt)]
          * dlmFunction(hfn, cosmp, sinmp, plm, l3, m3); //dlm[j];
      }
      // gij[lm2+lm1*kkri]=pi4*illp(lm2,lm1)*gij[lm2+lm1*kkri];
      devBgij[IDX(iOffset + lm2, jOffset + lm1, nrmat_ns)] = devBgij[IDX(iOffset + lm2, jOffset + lm1, nrmat_ns)]
        * pi4 * illp[IDX(lm2, lm1, ndlj_illp)];
    }

  }
}


__device__
void buildTmatNCuda(int ispin, int n_spin_pola, int n_spin_cant, int iie, int blkSizeTmatStore, int tmatStoreLDim,
                    int kkr1, int kkr2, int lizStoreIdx,
                    hipDoubleComplex *devTmatStore, int kkrsz_ns, hipDoubleComplex *tmat_n)
{
// Matrix<Complex> tmat_n(lsms.n_spin_cant*atom.kkrsz, lsms.n_spin_cant*atom.kkrsz);
  if(threadIdx.x == 0)
  {
    int im=0;
    if(n_spin_pola == n_spin_cant) // non polarized or spin canted
    {
      int kkrsz = kkrsz_ns/n_spin_cant;
      for(int js=0; js<n_spin_cant; js++)
      {
        int jsm = kkrsz*kkrsz_ns*js;
        for(int j=0; j<kkr1; j++)
        {
          for(int is=0; is<n_spin_cant; is++)
          {
            int jm=jsm+kkrsz_ns*j+kkrsz*is;
//                int one=1;
//                BLAS::zcopy_(&kkr1,&local.tmatStore(iie*local.blkSizeTmatStore+jm,atom.LIZStoreIdx[ir1]),&one,&tmat_n[im],&one);
            for(int i=0; i<kkr1; i++)
            {
              tmat_n[im+i] = devTmatStore[IDX(iie*blkSizeTmatStore+jm+i, lizStoreIdx, tmatStoreLDim)];
            }
            im+=kkr1;
          }
        }
      }
    } else { // spin polarized colinear version for ispin
      int kkrsz = kkrsz_ns/n_spin_cant;
      // int ispin=0;
      printf("warning: cant't test building kkrMatrix for collinear spin polarized yet!\n");
      // exit(1);
      int jsm = kkrsz*kkrsz*ispin; // copy spin up or down?
      for(int j=0; j<kkr1; j++)
      {
        int jm=jsm+kkrsz_ns*j;
//           int one=1;
//           BLAS::zcopy_(&kkr1,&local.tmatStore(iie*local.blkSizeTmatStore+jm,atom.LIZStoreIdx[ir1]),&one,&tmat_n[im],&one);
        for(int i=0; i<kkr1; i++)
        {
          tmat_n[im+i] = devTmatStore[IDX(iie*blkSizeTmatStore+jm+i, lizStoreIdx, tmatStoreLDim)];
        }
        im+=kkr1;
      }
    }
  }
  __syncthreads();
}

__global__
void buildKKRMatrixMultiplyKernelCuda(int *LIZlmax, int *LIZStoreIdx, int *offsets, int kkrsz_ns,
                                      int ispin, int n_spin_pola, int n_spin_cant, int iie, int blkSizeTmatStore, int tmatStoreLDim,
                                      hipDoubleComplex *devTmatStore, int nrmat_ns, hipDoubleComplex *devBgij, hipDoubleComplex *devM)
{
  int ir1 = blockIdx.x;
  int ir2 = blockIdx.y;
//  extern hipDoubleComplex __shared__ *tmat_n;
  hipDoubleComplex *tmat_n;
  int iOffset = offsets[ir1];
  int jOffset = offsets[ir2];

  if(ir1 != ir2)
  {
    int lmax1 = LIZlmax[ir1];
    int lmax2 = LIZlmax[ir2];
    int kkr1=(lmax1+1)*(lmax1+1);
    int kkr2=(lmax2+1)*(lmax2+1);
    int kkr1_ns = kkr1 * n_spin_cant;
    int kkr2_ns = kkr2 * n_spin_cant;

//        BLAS::zgemm_("n", "n", &kkr1_ns, &kkr2_ns, &kkr1_ns, &cmone,
//                     &local.tmatStore(iie*local.blkSizeTmatStore, devAtom.LIZStoreIdx[ir1]), &kkr1_ns,
//                     // &tmat_n(0, 0), &kkr1_ns,
//                     &bgij(iOffset, jOffset), &nrmat_ns, &czero,
//                     // &bgijSmall(0, 0), &kkrsz_ns, &czero,
//                     &m(iOffset, jOffset), &nrmat_ns);



//    for(int i=0; i<kkr1_ns; i++)
//      for(int j=0; j<kkr2_ns; j++)
//    buildTmatNCuda(ispin, n_spin_pola, n_spin_cant, iie, blkSizeTmatStore, tmatStoreLDim,
//                   kkr1, kkr2, LIZStoreIdx[ir1], devTmatStore, kkrsz_ns, tmat_n);

    int jsm =  kkrsz_ns * kkrsz_ns * ispin;

    tmat_n = &devTmatStore[IDX(iie*blkSizeTmatStore + jsm, LIZStoreIdx[ir1], tmatStoreLDim)];

    for(int ij=threadIdx.x; ij < kkr1_ns*kkr2_ns; ij += blockDim.x)
    {
      int i = ij % kkr1_ns;
      int j = ij / kkr1_ns;

      devM[IDX(iOffset + i, jOffset + j, nrmat_ns)] = make_hipDoubleComplex(0.0,0.0);
      for(int k=0; k<kkr1_ns ; k++)
        devM[IDX(iOffset + i, jOffset + j, nrmat_ns)] = devM[IDX(iOffset + i, jOffset + j, nrmat_ns)] -
          tmat_n[IDX(i,k,kkr1_ns)] * // tmat_n(i, k) * // local.tmatStore(iie*local.blkSizeTmatStore + , atom.LIZStoreIdx[ir1]) *
          devBgij[IDX(iOffset + k, jOffset + j, nrmat_ns)];
    }

  }
}


void buildKKRMatrixLMaxIdenticalCuda(LSMSSystemParameters &lsms, LocalTypeInfo &local, AtomData &atom,
                                     DeviceStorage &d, DeviceAtom &devAtom, int ispin,
                                     int iie, Complex energy, Complex prel, Complex *devM)
{
  hipblasHandle_t cublasHandle = DeviceStorage::getCublasHandle();
  int nrmat_ns = lsms.n_spin_cant*atom.nrmat; // total size of the kkr matrix
  int kkrsz_ns = lsms.n_spin_cant*atom.kkrsz; // size of t00 block
  bool fullRelativity = false;
  if(lsms.relativity == full) fullRelativity = true;

  // Complex cmone = Complex(-1.0,0.0);
  // Complex czero=0.0;

  Complex *devBgij = d.getDevBGij();
  // Matrix<Complex> bgijSmall(kkrsz_ns, kkrsz_ns);

  hipDoubleComplex cuEnergy = make_hipDoubleComplex(energy.real(), energy.imag());
  hipDoubleComplex cuPrel = make_hipDoubleComplex(prel.real(), prel.imag());

  unitMatrixCuda<Complex>(devM, nrmat_ns, nrmat_ns);
  zeroMatrixCuda(devBgij, nrmat_ns, nrmat_ns);

// calculate Bgij
// reuse ipvt for offsets
  int *devOffsets = d.getDevIpvt();

  std::vector<int> offsets(devAtom.numLIZ);
  for(int ir = 0; ir < devAtom.numLIZ; ir++)
    offsets[ir] = ir * kkrsz_ns;

  hipMemcpy(devOffsets, &offsets[0], atom.numLIZ*sizeof(int), hipMemcpyHostToDevice);

  size_t hfnOffset, sinmpOffset, cosmpOffset, plmOffset, dlmOffset;
  size_t smSize = sharedMemoryBGijCuda(lsms, &hfnOffset, &sinmpOffset, &cosmpOffset,
                                       &plmOffset, &dlmOffset);
#ifdef COMPARE_ORIGINAL
  printf("smSize = %zu\n", smSize);
  printf("  hfnOffset = %zu\n", hfnOffset);
  printf("  sinmpOffset = %zu\n", sinmpOffset);
  printf("  cosmpOffset = %zu\n", cosmpOffset);
  printf("  plmOffset = %zu\n", plmOffset);
  printf("  dlmOffset = %zu\n", dlmOffset);
  char *devTestSM;
  hipMalloc(&devTestSM, smSize);
    {
// test 
//  Matrix<Real> testLIZPos(3,atom.numLIZ);
//  Matrix<Complex> bgij(nrmat_ns, nrmat_ns);
  Complex testIlp1[2*lsms.maxlmax + 1];
//  hipMemcpy(&bgij[0], devBgij, nrmat_ns*nrmat_ns*sizeof(Complex), hipMemcpyDeviceToHost);
//  hipMemcpy(&testLIZPos[0], devAtom.LIZPos, 3*atom.numLIZ*sizeof(Real), hipMemcpyDeviceToHost);
  hipMemcpy(&testIlp1[0], DeviceConstants::ilp1, (2*lsms.maxlmax + 1)*sizeof(Complex), hipMemcpyDeviceToHost);
  printf("in calculateTauMatrix: before buildGijCudaKernel:\n");
  for(int l=0; l<2*lsms.maxlmax; l++)
  {
    printf("l=%d : ilp1 [%g + %gi] | DeviceConstats::ilp1 [%g + %gi]\n",l,IFactors::ilp1[l].real(),IFactors::ilp1[l].imag(), testIlp1[l].real(), testIlp1[l].imag());
  }
  }
#endif
  int threads = 256;
  // int threads = 1;
  dim3 blocks = dim3(devAtom.numLIZ, devAtom.numLIZ,1);
  buildGijCudaKernel<<<blocks,threads,smSize>>>(devAtom.LIZPos, devAtom.LIZlmax,
                                                DeviceConstants::lofk, DeviceConstants::mofk, DeviceConstants::ilp1, DeviceConstants::illp, DeviceConstants::cgnt,
                                                DeviceConstants::ndlj_illp, DeviceConstants::lmaxp1_cgnt, DeviceConstants::ndlj_cgnt,
                                                hfnOffset, sinmpOffset, cosmpOffset, plmOffset, dlmOffset,
                                                cuEnergy, cuPrel,
#if !defined(COMPARE_ORIGINAL)
                                                devOffsets, nrmat_ns, (hipDoubleComplex *)devBgij);
#else
                                                devOffsets, nrmat_ns, (hipDoubleComplex *)devBgij, devTestSM);

    {
// test
//  Matrix<Real> testLIZPos(3,atom.numLIZ);
//  Matrix<Complex> bgij(nrmat_ns, nrmat_ns);
  Complex testIlp1[2*lsms.maxlmax + 1];
//  hipMemcpy(&bgij[0], devBgij, nrmat_ns*nrmat_ns*sizeof(Complex), hipMemcpyDeviceToHost);
//  hipMemcpy(&testLIZPos[0], devAtom.LIZPos, 3*atom.numLIZ*sizeof(Real), hipMemcpyDeviceToHost);
  hipMemcpy(&testIlp1[0], DeviceConstants::ilp1, (2*lsms.maxlmax + 1)*sizeof(Complex), hipMemcpyDeviceToHost);
  printf("in calculateTauMatrix: before setBGijCuda:\n");
  for(int l=0; l<2*lsms.maxlmax; l++)
  {
    printf("l=%d : ilp1 [%g + %gi] | DeviceConstats::ilp1 [%g + %gi]\n",l,IFactors::ilp1[l].real(),IFactors::ilp1[l].imag(), testIlp1[l].real(), testIlp1[l].imag());
  }
  }
#endif

  setBGijCuda<<<blocks, threads>>>(fullRelativity, lsms.n_spin_cant, devAtom.LIZlmax,
                                   devOffsets, nrmat_ns, (hipDoubleComplex *)devBgij);

#ifdef COMPARE_ORIGINAL
  bool exitCompare = false;
  Matrix<Real> testLIZPos(3,atom.numLIZ);
  Matrix<Complex> bgij(nrmat_ns, nrmat_ns);
  Complex testIlp1[2*lsms.maxlmax + 1];
  hipMemcpy(&bgij[0], devBgij, nrmat_ns*nrmat_ns*sizeof(Complex), hipMemcpyDeviceToHost);
  hipMemcpy(&testLIZPos[0], devAtom.LIZPos, 3*atom.numLIZ*sizeof(Real), hipMemcpyDeviceToHost);
  hipMemcpy(&testIlp1[0], DeviceConstants::ilp1, (2*lsms.maxlmax + 1)*sizeof(Complex), hipMemcpyDeviceToHost);  

  for(int l=0; l<2*lsms.maxlmax; l++)
  {
    printf("l=%d : ilp1 [%g + %gi] | DeviceConstats::ilp1 [%g + %gi]\n",l,IFactors::ilp1[l].real(),IFactors::ilp1[l].imag(), testIlp1[l].real(), testIlp1[l].imag());
  }

  Complex testHfn[2*lsms.maxlmax + 1];
  Real testSinmp[2*lsms.maxlmax + 1];
  Real testCosmp[2*lsms.maxlmax + 1];
  // Real plm[((lsms.maxlmax+1) * (lsms.maxlmax+2)) / 2];
  Real testPlm[lsms.angularMomentumIndices.ndlm];
  Complex testDlm[lsms.angularMomentumIndices.ndlj];
  hipMemcpy(testHfn, devTestSM + hfnOffset, (2*lsms.maxlmax + 1)*sizeof(Complex), hipMemcpyDeviceToHost);
  hipMemcpy(testSinmp, devTestSM + sinmpOffset, (2*lsms.maxlmax + 1)*sizeof(Real), hipMemcpyDeviceToHost);
  hipMemcpy(testCosmp, devTestSM + cosmpOffset, (2*lsms.maxlmax + 1)*sizeof(Real), hipMemcpyDeviceToHost);
  hipMemcpy(testPlm, devTestSM + plmOffset, lsms.angularMomentumIndices.ndlm*sizeof(Real), hipMemcpyDeviceToHost);
  hipMemcpy(testDlm, devTestSM + dlmOffset, lsms.angularMomentumIndices.ndlj*sizeof(Complex), hipMemcpyDeviceToHost);

  for(int i = 0; i < atom.numLIZ; i++)
  {
    if(atom.LIZPos(0,i) != testLIZPos(0,i) ||
       atom.LIZPos(1,i) != testLIZPos(1,i) ||
       atom.LIZPos(2,i) != testLIZPos(2,i))
    {
      printf("atom.LIZPos(*,%d) [%lf,%lf,%lf] != devAtom.LIZPos(*,%d) [%lf,%lf,%lf]\n",
             i,atom.LIZPos(0,i),atom.LIZPos(1,i),atom.LIZPos(2,i),
             i,testLIZPos(0,i),testLIZPos(1,i),testLIZPos(2,i));
    }
  }
  // loop over the LIZ blocks
  Complex hfn[2*lsms.maxlmax + 1];
  Real sinmp[2*lsms.maxlmax + 1];
  Real cosmp[2*lsms.maxlmax + 1];
  // Real plm[((lsms.maxlmax+1) * (lsms.maxlmax+2)) / 2];
  Real plm[lsms.angularMomentumIndices.ndlm];
  Complex dlm[lsms.angularMomentumIndices.ndlj];
  Real rij[3];
  Real pi4=4.0*2.0*std::asin(1.0);
  for(int ir1 = 0; ir1 < atom.numLIZ; ir1++)
  {
    int iOffset = ir1 * kkrsz_ns; // this assumes that there are NO lStep reductions of lmax!!!
    for(int ir2 = 0; ir2 < atom.numLIZ; ir2++)
    {
      int jOffset = ir2 * kkrsz_ns; // this assumes that there are NO lStep reductions of lmax!!
      int lmax1 = atom.LIZlmax[ir1];
      int lmax2 = atom.LIZlmax[ir2];
      int kkri=(lmax1+1)*(lmax1+1);
      int kkrj=(lmax2+1)*(lmax2+1);
      rij[0]=atom.LIZPos(0,ir1)-atom.LIZPos(0,ir2);
      rij[1]=atom.LIZPos(1,ir1)-atom.LIZPos(1,ir2);
      rij[2]=atom.LIZPos(2,ir1)-atom.LIZPos(2,ir2);
      if(ir1 != ir2)
      {
        int kkr1 = kkri;
        int kkr2 = kkrj;
        Matrix<Complex> gijTest(kkr1,kkr2);
        Matrix<Complex> bgijTest(2*kkr1, 2*kkr2);
        int lmax=lsms.maxlmax;
        int kkrsz=(lmax+1)*(lmax+1);
        makegij_(&atom.LIZlmax[ir1],&kkr1,&atom.LIZlmax[ir2],&kkr2,
                 &lsms.maxlmax,&kkrsz,&lsms.angularMomentumIndices.ndlj,&lsms.angularMomentumIndices.ndlm,
                 &prel,&rij[0],&sinmp[0],&cosmp[0],
                 &sphericalHarmonicsCoeficients.clm[0],&plm[0],
                 &gauntCoeficients.cgnt(0,0,0),&gauntCoeficients.lmax,
                 &lsms.angularMomentumIndices.lofk[0],&lsms.angularMomentumIndices.mofk[0],
                 &iFactors.ilp1[0],&iFactors.illp(0,0),
                 &hfn[0],&dlm[0],&gijTest(0,0),
                 &pi4,&lsms.global.iprint,lsms.global.istop,32);

        if(ir1 == 0 && ir2 == 1)
        {
          for(int l=0; l<=atom.LIZlmax[ir1]+atom.LIZlmax[ir2]; l++)
          {
            if(sinmp[l] != testSinmp[l])
              printf("sinmp[%d] (%g) != testSinmp[%d] (%g)\n", l, sinmp[l], l, testSinmp[l]);
            if(cosmp[l] != testCosmp[l])
              printf("cosmp[%d] (%g) != testCosmp[%d] (%g)\n", l, cosmp[l], l, testCosmp[l]);
            if(hfn[l] != testHfn[l])
              printf("hfn[%d] (%g + %gi) != testHfn[%d] (%g + %gi)\n", l, hfn[l].real(), hfn[l].imag(), l, testHfn[l].real(), testHfn[l].imag());
          }
        }

        int idx=0;
        for(int i=0; i<kkri; i++)
          for(int j=0; j<kkrj; j++)
          {
            if(bgij(iOffset + i, jOffset + j) != gijTest(i,j))
              // if(bgij[idx] != gijTest[idx])
            {
              printf("buildBGijCPU [idx=%d]: bgij(%d + %d, %d + %d) [%g + %gi] != gijTest(%d, %d) [%g + %gi]\n", idx,
                     iOffset, i, jOffset, j, bgij(iOffset + i, jOffset + j).real(), bgij(iOffset + i, jOffset + j).imag(),
                     i, j, gijTest(i,j).real(), gijTest(i,j).imag());
              exitCompare = true;
            }
            if(bgij(iOffset + kkri + i, jOffset + kkrj + j) != gijTest(i,j))
              // if(bgij[idx] != gijTest[idx])
            {
              printf("buildBGijCPU : bgij(%d + %d, %d + %d) [%g + %gi] != gijTest(%d, %d) [%g + %gi]\n",
                     iOffset, i+kkri, jOffset, j+kkrj, bgij(iOffset + kkri + i, jOffset + kkrj + j).real(), bgij(iOffset + kkri + i, jOffset + kkrj + j).imag(),
                     i, j, gijTest(i,j).real(), gijTest(i,j).imag());
              exitCompare = true;
            }
            if(bgij(iOffset + kkri + i, jOffset + j) != 0.0) //gijTest(i+kkri,j))
              // if(bgij[idx] != gijTest[idx])
            {
              printf("buildBGijCPU : bgij(%d + %d, %d + %d) [%g + %gi] != 0.0\n",
                     iOffset, i+kkri, jOffset, j, bgij(iOffset + kkri + i, jOffset + j).real(), bgij(iOffset + kkri + i, jOffset + j).imag());
              exitCompare = true;
            }
            if(bgij(iOffset + i, jOffset + kkrj + j) != 0.0) //gijTest(i,j+kkrj))
              // if(bgij[idx] != gijTest[idx])
            {
              printf("buildBGijCPU : bgij(%d + %d, %d + %d) [%g + %gi] != 0.0\n",
                     iOffset, i, jOffset, j+kkrj, bgij(iOffset + i, jOffset + kkrj + j).real(), bgij(iOffset + i, jOffset + kkrj + j).imag());
              exitCompare = true;
            }
            idx++;
          }
      }
    }
  }

/*
  Complex psq=prel*prel;
  for(int ir1 = 0; ir1 < atom.numLIZ; ir1++)
  {
    int iOffset = ir1 * kkrsz_ns; // this assumes that there are NO lStep reductions of lmax!!!
    for(int ir2 = 0; ir2 < atom.numLIZ; ir2++)
    {
      int jOffset = ir2 * kkrsz_ns; // this assumes that there are NO lStep reductions of lmax!!
      int lmax1 = atom.LIZlmax[ir1];
      int lmax2 = atom.LIZlmax[ir2];
      int kkr1=(lmax1+1)*(lmax1+1);
      int kkr2=(lmax2+1)*(lmax2+1);
      int kkr1_ns = 2*kkr1;
      int kkr2_ns = 2*kkr2;
      int nrel_rel=0;
      if(lsms.relativity==full) nrel_rel=1;
      setgij_(&gijTest(0,0),&bgijTest(0,0),&kkr1,&kkr1_ns,&kkr2,&kkr2_ns,
              &lsms.n_spin_cant,&nrel_rel,&psq,&energy);
  idx=0;
  for(int i=0; i<2*kkri; i++)
    for(int j=0; j<2*kkrj; j++)
    {
      // if(bgij(iOffset + i, jOffset + j) != bgijTest(i,j))
      if(bgij[idx] != bgijTest[idx])
      {
        printf("buildBGijCPU  [idx=%d]: bgij(%d + %d, %d + %d) [%g + %gi] != bgijTest(%d, %d) [%g + %gi]\n", idx,
               iOffset, i, jOffset, j, bgij(iOffset + i, jOffset + j).real(), bgij(iOffset + i, jOffset + j).imag(),
               i, j, bgijTest(i,j).real(), bgijTest(i,j).imag());
        exitCompare = true;
      }
      idx++;
    }

  if((ir1==1 && ir2==0) || (ir1==10 && ir2==0))
  {
    printf("ir1=%d, ir2=%d: bgij(0,0) = %g + %gi; bgijTest(0,0) = %g + %gi\n",
           ir1, ir2, bgij(0,0).real(), bgij(0,0).imag(), bgijTest(0,0).real(), bgijTest(0,0).imag());
    printf("    rij = %g %g %g;  prel=%g + %gi\n", rij[0],  rij[1], rij[2], prel.real(), prel.imag());
    printf("    kkr1 = %d; kkr2 = %d; kkrsz = %d\n", kkr1, kkr2, kkrsz);
  }
*/

#endif

  smSize = kkrsz_ns*kkrsz_ns*sizeof(hipDoubleComplex);
  threads = 256;
  // threads = 1;
  // printf("buildKKRMatrixMultiplyKernelCuda: smSize=%zu\n",smSize);
  // note that the shared memory requiremets of the present implementation is too large for lmax>3
  // buildKKRMatrixMultiplyKernelCuda<<<blocks, threads, smSize>>>(devAtom.LIZlmax, devAtom.LIZStoreIdx, devOffsets,
  buildKKRMatrixMultiplyKernelCuda<<<blocks, threads>>>(devAtom.LIZlmax, devAtom.LIZStoreIdx, devOffsets,
                                                        kkrsz_ns, ispin, lsms.n_spin_pola, lsms.n_spin_cant,
                                                        iie, d.getBlkSizeTmatStore(), d.getTmatStoreLDim(),
                                                        (hipDoubleComplex *) d.getDevTmatStore(), nrmat_ns,
                                                        (hipDoubleComplex *) devBgij, (hipDoubleComplex *) devM);

#ifdef BUILDKKRMATRIX_CUDA_DEBUG
  std::ofstream myfile;
  auto filename = "M" + std::to_string(iie) + "_" + std::to_string(ispin);
  myfile.open(filename.c_str());
  for (int i = 0; i < nrmat_ns * nrmat_ns; i++) {
    Complex buffer;
    hipMemcpy(&buffer, &devM[i], sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
    myfile << std::real(buffer) << " " << std::imag(buffer) << std::endl;
  }
  myfile.close();
#endif

  /*
  // loop over the LIZ blocks
  for(int ir1 = 0; ir1 < devAtom.numLIZ; ir1++)
  {
    int iOffset = ir1 * kkrsz_ns; // this assumes that there are NO lStep reductions of lmax!!!
    for(int ir2 = 0; ir2 < devAtom.numLIZ; ir2++)
    {
      if(ir1 != ir2)
      {
        int jOffset = ir2 * kkrsz_ns; // this assumes that there are NO lStep reductions of lmax!!!
        int lmax1 = devAtom.LIZlmax[ir1];
        int lmax2 = devAtom.LIZlmax[ir2];
        int kkr1=(lmax1+1)*(lmax1+1);
        int kkr2=(lmax2+1)*(lmax2+1);
        int kkr1_ns = kkr1 * lsms.n_spin_cant;
        int kkr2_ns = kkr2 * lsms.n_spin_cant;
        
        // buildBGijCuda(lsms, atom, ir1, ir2, rij, energy, prel, iOffset, jOffset, bgij);
        // buildBGijCPU(lsms, atom, ir1, ir2, rij, energy, prel, 0, 0, bgijSmall);
             
        BLAS::zgemm_("n", "n", &kkr1_ns, &kkr2_ns, &kkr1_ns, &cmone,
                     &local.tmatStore(iie*local.blkSizeTmatStore, devAtom.LIZStoreIdx[ir1]), &kkr1_ns,
                     // &tmat_n(0, 0), &kkr1_ns,
                     &bgij(iOffset, jOffset), &nrmat_ns, &czero,
                     // &bgijSmall(0, 0), &kkrsz_ns, &czero,
                     &m(iOffset, jOffset), &nrmat_ns);
        
//        for(int i=0; i<kkr1_ns; i++)
//          for(int j=0; j<kkr2_ns; j++)
//          {
//            m(iOffset + i, jOffset + j) = 0.0;
//            for(int k=0; k<kkr1_ns ; k++)
//              m(iOffset + i, jOffset + j) -= tmat_n(i, k) * // local.tmatStore(iie*local.blkSizeTmatStore + , atom.LIZStoreIdx[ir1]) *
//                // bgij(iOffset + k, jOffset + j);
//                bgijSmall(k, j);
//          }
        
      }
    }
  }
  */
#ifdef COMPARE_ORIGINAL
  Matrix<Complex> mCPU(nrmat_ns,nrmat_ns);
  Matrix<Complex> mGPU(nrmat_ns,nrmat_ns);

  hipMemcpy(&mGPU(0,0), devM, nrmat_ns*nrmat_ns*sizeof(Complex), hipMemcpyDeviceToHost);
  buildKKRMatrixCPU(lsms, local, atom, iie, energy, prel, mCPU);

  for(int i=0; i<nrmat_ns; i++)
    for(int j=0; j<nrmat_ns; j++)
        {
            if(mCPU(i,j) != mGPU(i,j))
              // if(bgij[idx] != gijTest[idx])
            {
              printf("buildBGijCPU : mCPU(%d, %d) [%g + %gi] != mGPU(%d, %d) [%g + %gi]\n",
                     i, j, mCPU(i, j).real(), mCPU(i, j).imag(),
                     i, j, mGPU(i,j).real(), mGPU(i,j).imag());
              exitCompare = true;
            }
        }

  if(exitCompare)
    exit(1);
#endif

}
void buildKKRMatrixLMaxDifferentCuda(LSMSSystemParameters &lsms, LocalTypeInfo &local, AtomData &atom,
                                     DeviceStorage &d, DeviceAtom &devAtom, int ispin,
                                     int iie, Complex energy, Complex prel, Complex *devM)
{
  hipblasHandle_t cublasHandle = DeviceStorage::getCublasHandle();
  int nrmat_ns = lsms.n_spin_cant*atom.nrmat; // total size of the kkr matrix
  int kkrsz_ns = lsms.n_spin_cant*atom.kkrsz; // size of t00 block
  bool fullRelativity = false;
  if(lsms.relativity == full) fullRelativity = true;

  // Complex cmone = Complex(-1.0,0.0);
  // Complex czero=0.0;

  Complex *devBgij = d.getDevBGij();
  // Matrix<Complex> bgijSmall(kkrsz_ns, kkrsz_ns);

  hipDoubleComplex cuEnergy = make_hipDoubleComplex(energy.real(), energy.imag());
  hipDoubleComplex cuPrel = make_hipDoubleComplex(prel.real(), prel.imag());

  unitMatrixCuda<Complex>(devM, nrmat_ns, nrmat_ns);
  zeroMatrixCuda(devBgij, nrmat_ns, nrmat_ns);

// calculate Bgij
// reuse ipvt for offsets
  int *devOffsets = d.getDevIpvt();

  std::vector<int> offsets(devAtom.numLIZ);
  offsets[0] = 0;
  for(int ir = 1; ir < atom.numLIZ; ir++)
    offsets[ir] = offsets[ir-1] + lsms.n_spin_cant * (atom.LIZlmax[ir-1]+1)*(atom.LIZlmax[ir-1]+1);

  hipMemcpy(devOffsets, &offsets[0], atom.numLIZ*sizeof(int), hipMemcpyHostToDevice);

  size_t hfnOffset, sinmpOffset, cosmpOffset, plmOffset, dlmOffset;
  size_t smSize = sharedMemoryBGijCuda(lsms, &hfnOffset, &sinmpOffset, &cosmpOffset,
                                       &plmOffset, &dlmOffset);
#ifdef COMPARE_ORIGINAL
  char *devTestSM;
  hipMalloc(&devTestSM, smSize);
#endif
  int threads = 256;
  dim3 blocks = dim3(devAtom.numLIZ, devAtom.numLIZ,1);
  buildGijCudaKernel<<<blocks,threads,smSize>>>(devAtom.LIZPos, devAtom.LIZlmax,
                                                DeviceConstants::lofk, DeviceConstants::mofk, DeviceConstants::ilp1, DeviceConstants::illp, DeviceConstants::cgnt,
                                                DeviceConstants::ndlj_illp, DeviceConstants::lmaxp1_cgnt, DeviceConstants::ndlj_cgnt,
                                                hfnOffset, sinmpOffset, cosmpOffset, plmOffset, dlmOffset,
                                                cuEnergy, cuPrel,
#if !defined(COMPARE_ORIGINAL)
                                                devOffsets, nrmat_ns, (hipDoubleComplex *)devBgij);
#else
                                                devOffsets, nrmat_ns, (hipDoubleComplex *)devBgij, devTestSM);
#endif
  setBGijCuda<<<blocks, threads>>>(fullRelativity, lsms.n_spin_cant, devAtom.LIZlmax,
                                   devOffsets, nrmat_ns, (hipDoubleComplex *)devBgij);


  smSize = kkrsz_ns*kkrsz_ns*sizeof(hipDoubleComplex);
  threads = 256;
  // threads = 1;
  // printf("buildKKRMatrixMultiplyKernelCuda: smSize=%zu\n",smSize);
  // note that the shared memory requiremets of the present implementation is too large for lmax>3
  // buildKKRMatrixMultiplyKernelCuda<<<blocks, threads, smSize>>>(devAtom.LIZlmax, devAtom.LIZStoreIdx, devOffsets,
  buildKKRMatrixMultiplyKernelCuda<<<blocks, threads>>>(devAtom.LIZlmax, devAtom.LIZStoreIdx, devOffsets,
                                                        kkrsz_ns, ispin, lsms.n_spin_pola, lsms.n_spin_cant,
                                                        iie, d.getBlkSizeTmatStore(), d.getTmatStoreLDim(),
                                                        (hipDoubleComplex *)d.getDevTmatStore(), nrmat_ns,
                                                        (hipDoubleComplex *)devBgij, (hipDoubleComplex *)devM);
  /* 
  // loop over the LIZ blocks
  for(int ir1 = 0; ir1 < devAtom.numLIZ; ir1++)
  {
    int iOffset = ir1 * kkrsz_ns; // this assumes that there are NO lStep reductions of lmax!!!
    for(int ir2 = 0; ir2 < devAtom.numLIZ; ir2++)
    {
      if(ir1 != ir2)
      {
        int jOffset = ir2 * kkrsz_ns; // this assumes that there are NO lStep reductions of lmax!!!
        int lmax1 = devAtom.LIZlmax[ir1];
        int lmax2 = devAtom.LIZlmax[ir2];
        int kkr1=(lmax1+1)*(lmax1+1);
        int kkr2=(lmax2+1)*(lmax2+1);
        int kkr1_ns = kkr1 * lsms.n_spin_cant;
        int kkr2_ns = kkr2 * lsms.n_spin_cant;
        
        // buildBGijCuda(lsms, atom, ir1, ir2, rij, energy, prel, iOffset, jOffset, bgij);
        // buildBGijCPU(lsms, atom, ir1, ir2, rij, energy, prel, 0, 0, bgijSmall);
             
        BLAS::zgemm_("n", "n", &kkr1_ns, &kkr2_ns, &kkr1_ns, &cmone,
                     &local.tmatStore(iie*local.blkSizeTmatStore, devAtom.LIZStoreIdx[ir1]), &kkr1_ns,
                     // &tmat_n(0, 0), &kkr1_ns,
                     &bgij(iOffset, jOffset), &nrmat_ns, &czero,
                     // &bgijSmall(0, 0), &kkrsz_ns, &czero,
                     &m(iOffset, jOffset), &nrmat_ns);
        
//        for(int i=0; i<kkr1_ns; i++)
//          for(int j=0; j<kkr2_ns; j++)
//          {
//            m(iOffset + i, jOffset + j) = 0.0;
//            for(int k=0; k<kkr1_ns ; k++)
//              m(iOffset + i, jOffset + j) -= tmat_n(i, k) * // local.tmatStore(iie*local.blkSizeTmatStore + , atom.LIZStoreIdx[ir1]) *
//                // bgij(iOffset + k, jOffset + j);
//                bgijSmall(k, j);
//          }
        
      }
    }
  }
  */
}



void buildKKRMatrixCuda(LSMSSystemParameters &lsms, LocalTypeInfo &local, AtomData &atom,
                        DeviceStorage &devStorage, DeviceAtom &devAtom, int ispin,
                        int iie, Complex energy, Complex prel, Complex *devM)
{
  // decide between identical lmax and different lmax:

  // printf("buildKKRMatrixCuda not finished yet!\n");
  // exit(1);

  bool lmaxIdentical = true;

  if(atom.LIZlmax[0] != lsms.maxlmax)
  {
    lmaxIdentical = false;
    printf("atom.LIZlmax[0] (=%d) != lsms.maxlmax (=%d)\n",atom.LIZlmax[0], lsms.maxlmax);
  }
  for(int ir = 0; ir < atom.numLIZ; ir++)
  {
    if(atom.LIZlmax[ir] != atom.LIZlmax[0])
      lmaxIdentical = false;
  }

  if(lmaxIdentical)
  {
    // printf("lmax identical in buildKKRMatrix\n");

    buildKKRMatrixLMaxIdenticalCuda(lsms, local, atom, devStorage, devAtom, ispin,
                                    iie, energy, prel, devM);
  } else {
    // printf("lmax not identical in buildKKRMatrix\n");
    buildKKRMatrixLMaxDifferentCuda(lsms, local, atom, devStorage, devAtom, ispin,
                                    iie, energy, prel, devM);
  }
}
