#include "hip/hip_runtime.h"
/* -*- mode: C++; c-file-style: "bsd"; c-basic-offset: 2; indent-tabs-mode: nil -*- */

// test the inverion algorithm for multiple scattering codes for the solution of
// tau = (1 - tG)^-1 t
// where t is a block diagonal matrix
// note that the diagonal blocks G_ii == 0

#include <stdio.h>

#include "Complex.hpp"
#include "Matrix.hpp"
#include <vector>
#include <chrono>
#include <ctime>

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipblas.h>
#include <hipsolver.h>

#include "inversionTest_cuda.hpp"

/*
class DeviceData
{
public:
  std::vector<hipDoubleComplex *> tMatrices;
  hipDoubleComplex *tMatrixStore;
  hipDoubleComplex *tau;
  hipDoubleComplex *tau00;
  hipDoubleComplex *m;
  hipDoubleComplex *G0;
  int *ipiv;
  int *info;
};
*/

/*
class DeviceHandles
{
  hipblasHandle_t cublasHandle;
  hipsolverHandle_t cusolverHandle;
};
*/

void allocDeviceData(DeviceHandles &deviceHandles, DeviceData &d, int blockSize, int numBlocks)
{
  int n=blockSize*numBlocks;
  hipMalloc((void**)&d.m, n*n*sizeof(hipDoubleComplex));
  hipMalloc((void**)&d.G0, n*n*sizeof(hipDoubleComplex));
  hipMalloc((void**)&d.tau, n*blockSize*sizeof(hipDoubleComplex));
  hipMalloc((void**)&d.t, n*blockSize*sizeof(hipDoubleComplex));
  hipMalloc((void**)&d.tau00, blockSize*blockSize*sizeof(hipDoubleComplex));
  hipMalloc((void**)&d.tMatrixStore, blockSize*blockSize*numBlocks*sizeof(hipDoubleComplex));
  d.tMatrices.resize(numBlocks);
  for(int i=0; i<numBlocks; i++)
    d.tMatrices[i] = &d.tMatrixStore[blockSize * blockSize * i];
  hipMalloc((void**)&d.ipiv, n*sizeof(int));
  hipMalloc((void**)&d.info, sizeof(int));

  int lWork;
  hipsolverDnZgetrf_bufferSize(deviceHandles.cusolverDnHandle, n, n,
    d.m, n, &lWork);

  d.workBytes = 0;
#ifndef ARCH_IBM
 hipsolverDnZZgesv_bufferSize(deviceHandles.cusolverDnHandle, n, blockSize,
    d.m, n, d.ipiv, d.t, n, d.tau, n,
    d.work, &d.workBytes);
#endif

  d.workBytes = std::max(d.workBytes*sizeof(hipDoubleComplex),
        lWork*sizeof(hipDoubleComplex));
  hipMalloc((void**)&d.work, d.workBytes);
}

void freeDeviceData(DeviceData &d)
{
  hipFree(d.tau);
  hipFree(d.tau00);
  hipFree(d.m);
  hipFree(d.G0);
  hipFree(d.tMatrixStore);
  hipFree(d.ipiv);
  hipFree(d.info);
  hipFree(d.t);
  hipFree(d.work);
}

// #include "makegij_new.cpp"

void usage_cuda(const char *name)
{
  printf("usage: %s <matrix type> [options]\n",name);
  printf("  matrix type: 1: 1-tG and G, t Hilbert matrices, options: <block size> <num blocks>\n");
  printf("               2: 1-tG and G = -1, t = 1\n");
}

#define IDX(i, j, lDim) (((j)*(lDim))+(i))

template <typename T>
void zeroMatrixCuda(T *devM, int lDim, int nCol)
{
//  for(int i=0; i<m.n_row(); i++)
//    for(int j=0; j<m.n_col(); j++)
//      m(i,j) = 0.0;
  hipMemset(devM, 0, lDim*nCol*sizeof(T));
}

template <typename T>
__global__ void setDiagonalKernel(T *devM, int lDim, int nCol, T val)
{
  int i=blockIdx.x;
  if(i<nCol)
  {
    devM[IDX(i, i, lDim)] = val;
  }
}

template <typename T>
__global__ void addDiagonalKernel(T *devM, int lDim, int nCol, T val)
{
  int i=blockIdx.x;
  if(i<nCol)
  {
    devM[IDX(i, i, lDim)] = hipCadd(devM[IDX(i, i, lDim)], val);
  }
}

template <typename T>
void unitMatrixCuda(T *devM, int lDim, int nCol)
{
  zeroMatrixCuda(devM, lDim, nCol);
  setDiagonalKernel<<<nCol,1>>>(devM, lDim, nCol, 1.0);
}

/*
Real matrixDistance(Matrix<Complex> &a, Matrix<Complex> &b)
{
  Real d;

  for(int i=0; i<a.n_col(); i++)
    for(int j=0; j<a.n_row(); j++)
      d += ((a(i,j)-b(i,j)) * std::conj(a(i,j)-b(i,j))).real();
  
  return std::sqrt(d);
}
*/

template <typename T>
__global__ void makeHilbertMatrixKernel(T *devM, int lDim, int nCol)
{
  int i = blockIdx.x;
  if(i<lDim)
  {
    for(int j=0; j<nCol; j++)
      devM[IDX(i,j, lDim)] = 1.0/(Complex(i+j+1));
  }
}

template <typename T>
__global__ void zeroDiagonalBlocksKernel(T *devM, int lDim, int nCol, int blockSize)
{
  int iBlock = blockIdx.x;
  int jBlock = blockIdx.y;
  if(iBlock<nCol/blockSize)
    if(jBlock<nCol/blockSize)
    {
      int ii=iBlock*blockSize;
      int jj=jBlock*blockSize;
      for(int i=0; i<std::min(blockSize, nCol-ii); i++)
        for(int j=0; j<std::min(blockSize, nCol-jj); j++)
          devM[IDX(ii+i, jj+j, lDim)] = 0.0;
    }
}

template <typename T>
void zeroDiagonalBlocksCuda(T *devM, int lDim, int nCol, int blockSize)
{
  zeroDiagonalBlocksKernel<<<nCol/blockSize,nCol/blockSize>>>(devM, lDim, nCol, blockSize);
}

void makeType1MatrixGPU(hipblasHandle_t cublasHandle, DeviceData &d, int blockSize, int numBlocks)
{
  int n=blockSize*numBlocks;
  hipDoubleComplex one = {1.0, 0.0};
  hipDoubleComplex mone = {-1.0, 0.0};
  hipDoubleComplex zero = {0.0, 0.0};
  std::vector<hipDoubleComplex *> ts(numBlocks*numBlocks);
  std::vector<hipDoubleComplex *> G0s(numBlocks*numBlocks);
  std::vector<hipDoubleComplex *> ms(numBlocks*numBlocks);

  for(int iBlock=0; iBlock<numBlocks; iBlock++)
  {
    for(int jBlock=0; jBlock<numBlocks; jBlock++)
    {
      ts[iBlock + jBlock*numBlocks] = d.tMatrices[iBlock];
      G0s[iBlock + jBlock*numBlocks] = &d.G0[IDX(iBlock*blockSize,jBlock*blockSize,n)];
      ms[iBlock + jBlock*numBlocks] = &d.m[IDX(iBlock*blockSize,jBlock*blockSize,n)];
    }
  }

  hipblasZgemmBatched(cublasHandle, HIPBLAS_OP_N, HIPBLAS_OP_N, blockSize, blockSize, blockSize, &mone,
                                  &ts[0], blockSize,
                                  &G0s[0], n,
                                  &zero,
                                  &ms[0], n, 
                                  numBlocks*numBlocks);
  addDiagonalKernel<<<n,1>>>(d.m, n, n, one);
}

void transferMatrixToGPU(hipDoubleComplex *devM, Matrix<Complex> &m)
{
  hipMemcpy(devM, &m(0,0), m.l_dim()*m.n_col()*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
}

void transferMatrixFromGPU(Matrix<Complex> &m, hipDoubleComplex *devM)
{
  hipMemcpy(&m(0,0), devM,  m.l_dim()*m.n_col()*sizeof(hipDoubleComplex), hipMemcpyDeviceToHost);
}

__global__ void copyTMatrixToTau(hipDoubleComplex *tau, hipDoubleComplex *t, int blockSize, int numBlocks)
{
  int i = blockIdx.x;
  int n = blockSize*numBlocks;
  if(i < blockSize)
  {
    for(int j=0; j<blockSize; j++)
      tau[IDX(i,j,n)] = t[IDX(i,j,blockSize)];
  }
}

__global__ void copyTauToTau00(hipDoubleComplex *tau00, hipDoubleComplex *tau, int blockSize, int numBlocks)
{
  int i = blockIdx.x;
  int n = blockSize*numBlocks;
  if(i < blockSize)
  {
    for(int j=0; j<blockSize; j++)
      tau00[IDX(i,j,blockSize)] = tau[IDX(i,j,n)];
  }
}

void solveTau00zgetrf_cublas(hipblasHandle_t cublasHandle, DeviceData &d,
                             Matrix<Complex> &tau00, int blockSize, int numBlocks)
{
  // reference algorithm. Use LU factorization and linear solve for dense matrices in LAPACK
  hipDoubleComplex *Aarray[1], *Barray[1];
 
  // printf("zero Matrix\n"); 
  zeroMatrixCuda(d.tau, blockSize*numBlocks, blockSize);
  // printf("copyTMatrixToTau\n");
  copyTMatrixToTau<<<blockSize,1>>>(d.tau, d.tMatrices[0], blockSize, numBlocks);

  Barray[0] = d.tau;

  Aarray[0] = d.m;
  
  int n = blockSize * numBlocks;
  int *ipivArray=d.ipiv;
  int *infoArray=d.info;
  int info;

  // printf("hipblasZgetrfBatched\n");
  hipblasZgetrfBatched(cublasHandle, n, Aarray, n, ipivArray, infoArray, 1);
  // printf("hipblasZgetrsBatched\n");

  hipblasZgetrsBatched(cublasHandle, HIPBLAS_OP_N, n, blockSize, Aarray, n, ipivArray,
                      Barray, n, &info, 1);

  // copy result into tau00
  // printf("copyTauToTau00\n");
  copyTauToTau00<<<blockSize,1>>>(d.tau00, d.tau, blockSize, numBlocks);
  // printf("transferMatrixFromGPU\n");
  transferMatrixFromGPU(tau00, d.tau00);
}

template <typename T>
void unitMatrix(Matrix<T> &m)
{
  for(int i=0; i<m.n_row(); i++)
  {
    for(int j=0;j<m.n_col(); j++)
      m(i,j) = 0.0;
    m(i,i) = 1.0;
  }
}

void block_inverse_cublas(hipblasHandle_t handle, Matrix<Complex> &a, int *blk_sz, int nblk, Matrix<Complex> &delta, int *ipvt, int *idcol, DeviceData &devD);

void solveTau00zblocklu_cublas(hipblasHandle_t handle, DeviceData &devData, Matrix<Complex> &tau00, Matrix<Complex> &m, std::vector<Matrix<Complex> > &tMatrices, int blockSize, int numBlocks)
{
  int nrmat_ns = blockSize * numBlocks;
  int ipvt[nrmat_ns];
  int info;
  int nblk = 3;
  Matrix<Complex> delta(blockSize, blockSize);

  int blk_sz[1000];

  blk_sz[0]=blockSize;
  if(nblk==1)
    blk_sz[0]=nrmat_ns;
  else if(nblk==2)
    blk_sz[1]=nrmat_ns-blk_sz[0];
  else if(nblk>2)
  {
    int min_sz=(nrmat_ns-blk_sz[0])/(nblk-1);
    int rem=(nrmat_ns-blk_sz[0])%(nblk-1);
    int i=1;
    for(;i<=rem;i++)
      blk_sz[i]=min_sz+1;
    for(;i<nblk;i++)
      blk_sz[i]=min_sz;
  }

  int idcol[blk_sz[0]]; idcol[0]=0;

  // with m = [[A B][C D]], A: blk_sz[0] x blk_sz[0]
  // calculate the Schur complement m/D of m with A set to zero,
  // i.e. delta = B D^-1 C
  block_inverse_cublas(handle, m, blk_sz, nblk, delta, ipvt, idcol, devData);


  Matrix<Complex> wbig(blockSize, blockSize);
// setup unit matrix...............................................
// n.b. this is the top diagonal block of the kkr matrix m
//      i.e. 1 - t_0 G_00, with G_ii == 0 this is just the unit matrix

  unitMatrix(wbig);

// c     get 1-delta and put it in wbig

  for(int i=0; i<blockSize; i++)
    for(int j=0; j<blockSize; j++)
      wbig(i,j) -= delta(i,j);
//  c     ================================================================
// c     create tau00 => {[1-t*G]**(-1)}*t : for central site only.......
// c     ----------------------------------------------------------------

  LAPACK::zgetrf_(&blockSize, &blockSize, &wbig(0,0), &blockSize, ipvt, &info);

  for(int i=0; i<blockSize; i++)
    for(int j=0; j<blockSize; j++)
      tau00(i,j) = tMatrices[0](i,j);

  LAPACK::zgetrs_("N", &blockSize, &blockSize, &wbig(0,0), &blockSize, ipvt, &tau00(0,0), &blockSize, &info);

}


#ifndef ARCH_IBM
void solveTau00zzgesv_cusolver(DeviceHandles &deviceHandles, DeviceData &deviceData, Matrix<Complex> &tau00, Matrix<Complex> &m, std::vector<Matrix<Complex> > &tMatrices, int blockSize, int numBlocks)
{
  // reference algorithm. Use LU factorization and linear solve for dense matrices in LAPACK

  zeroMatrixCuda(deviceData.tau, blockSize*numBlocks, blockSize);
  zeroMatrixCuda(deviceData.t, blockSize*numBlocks, blockSize);
  copyTMatrixToTau<<<blockSize,1>>>(deviceData.t, deviceData.tMatrices[0], blockSize, numBlocks);

  int n = blockSize * numBlocks;
  int info, iter;

  hipsolverStatus_t status = hipsolverDnZZgesv(deviceHandles.cusolverDnHandle, n, blockSize,
    deviceData.m, n, deviceData.ipiv, deviceData.t, n, deviceData.tau, n,
    deviceData.work, deviceData.workBytes, &iter, deviceData.info);

  if(status!=HIPSOLVER_STATUS_SUCCESS)
  {
    printf("hipsolverDnZZgesv returned %d\n",status);
  }

  copyTauToTau00<<<blockSize,1>>>(deviceData.tau00, deviceData.tau, blockSize, numBlocks);
  transferMatrixFromGPU(tau00, deviceData.tau00);
}
#endif

void solveTau00zgetrf_cusolver(DeviceHandles &deviceHandles, DeviceData &deviceData,
     Matrix<Complex> &tau00, int blockSize, int numBlocks)
{
  int n = blockSize * numBlocks;
  // reference algorithm. Use LU factorization and linear solve for dense matrices in LAPACK
  Matrix<Complex> tau(blockSize * numBlocks, blockSize);

  zeroMatrixCuda(deviceData.tau, n, blockSize);
  copyTMatrixToTau<<<blockSize,1>>>(deviceData.tau, deviceData.tMatrices[0], blockSize, numBlocks);

  hipsolverDnZgetrf(deviceHandles.cusolverDnHandle, n, n, 
           deviceData.m, n,
           (hipDoubleComplex *)deviceData.work,
           deviceData.ipiv,
           deviceData.info );
  // LAPACK::zgetrf_(&n, &n, &m(0,0), &n, &ipiv[0], &info);

  hipsolverDnZgetrs(deviceHandles.cusolverDnHandle, HIPBLAS_OP_N, n, blockSize,
      deviceData.m, n, deviceData.ipiv, deviceData.tau, n, deviceData.info);
  // LAPACK::zgetrs_("N", &n, &blockSize, &m(0,0), &n, &ipiv[0], &tau(0,0), &n, &info);

  // copy result into tau00
  copyTauToTau00<<<blockSize,1>>>(deviceData.tau00, deviceData.tau, blockSize, numBlocks);
  transferMatrixFromGPU(tau00, deviceData.tau00);
}


void transferTest(DeviceHandles &deviceHandles, DeviceData &deviceData, Matrix<Complex> &tau00, int blockSize, int numBlocks)
{
  copyTMatrixToTau<<<blockSize,1>>>(deviceData.t, deviceData.tMatrices[0], blockSize, numBlocks);
  copyTauToTau00<<<blockSize,1>>>(deviceData.tau00, deviceData.t, blockSize, numBlocks);
  transferMatrixFromGPU(tau00, deviceData.tau00);
}

void initCuda(DeviceHandles &dh)
{
  hipblasCreate(&dh.cublasHandle);
  hipsolverDnCreate(&dh.cusolverDnHandle);
}

void finalizeCuda(DeviceHandles &dh)
{
  hipsolverDnDestroy(dh.cusolverDnHandle);
  hipblasDestroy(dh.cublasHandle);
}


