#include "hip/hip_runtime.h"
// -*- mode: c++; -*-

#include <stdlib.h>
#include "Real.hpp"
#include "Complex.hpp"
#include "Matrix.hpp"

#include "DeviceMatrix.hpp"
#include "DeviceArray3d.hpp"
#include "DeviceVector.hpp"
#include "Main/SystemParameters.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsolver.h>
#include <iostream>

#ifdef _OPENMP
#include <omp.h>
#else
#ifndef LSMS_DUMMY_OPENMP
#define LSMS_DUMMY_OPENMP
inline int omp_get_max_threads() {return 1;}
inline int omp_get_num_threads() {return 1;}
inline int omp_get_thread_num() {return 0;}
#endif
#endif

#include "DeviceStorage.hpp"

// #include "cuda_error.h"
#include "deviceCheckError.hpp"

using namespace std;

//TODO move inside DeviceStorage?
//allocate a thread specific matrix on the host and pin its memory
extern "C"
Complex *get_host_m_(const int &max_nrmat_ns) {
  static Complex *m_v = 0;
  static int cur_size = 0;
  static hipError_t pinned;

  if (cur_size < max_nrmat_ns) {

    //release previously allocated memory
    if (m_v != 0) {
      if (pinned) hipHostFree(m_v);
      else free(m_v);
    }

    //allocate new memory
    pinned = hipHostMalloc((void **) &m_v, max_nrmat_ns * max_nrmat_ns * sizeof(Complex) * omp_get_max_threads());

    if (pinned != hipSuccess) {
      fprintf(stderr, "Matrix not pinned\n");
      m_v = (Complex *) malloc(max_nrmat_ns * max_nrmat_ns * sizeof(Complex) * omp_get_max_threads());
    }
    cur_size = max_nrmat_ns;
  }
  return m_v;
}

/*
static const int MAX_THREADS=16;
class DeviceStorage {
private:
  static int nThreads;
  static Complex *dev_m[MAX_THREADS], *dev_bgij[MAX_THREADS], *dev_tmat_n[MAX_THREADS];
  static Complex *dev_tau[MAX_THREADS], *dev_tau00[MAX_THREADS];
  static int *dev_ipvt[MAX_THREADS];
  static hipblasHandle_t hipblas.h[MAX_THREADS];
  static hipsolverHandle_t cusolverDnHandle[MAX_THREADS];
  static hipEvent_t event[MAX_THREADS];
  static hipStream_t stream[MAX_THREADS][2];
  static size_t dev_workBytes[MAX_THREADS];
  static void *dev_work[MAX_THREADS];
  static DeviceMatrix<Complex> dev_tmat_store;
  static bool initialized;
public:
*/

int DeviceStorage::allocate(int kkrsz_max, int nspin, int numLIZ, int _nThreads) {
  if (!initialized) {
    //printf("*************************************MEMORY IS BEING ALLOCATED\n");
    if (_nThreads > MAX_THREADS) {
      printf("nThreads (%d) in DeviceStorage::allocate exceeds MAX_THREADS (%d)\n", _nThreads, MAX_THREADS);
      printf("  change MAX_THREADS in src/Accelerator/DeviceStorage.cu and recompile!\n");
      exit(1);
    }
    nThreads = _nThreads;
    int N = kkrsz_max * nspin * numLIZ;
    // printf("DeviceStorage::alocate N=%d\n",N);
    for (int i = 0; i < nThreads; i++) {
      hipError_t err;
      err = hipMalloc((void **) &dev_m[i], N * N * sizeof(Complex));
      if (err != hipSuccess) {
        printf("failed to allocate dev_m[%d], size=%d, err=%d\n",
               i, N * N * sizeof(Complex), err);
        exit(1);
      }
      hipMalloc((void **) &dev_ipvt[i], N * sizeof(int));
      hipMalloc((void **) &dev_info[i], nThreads * sizeof(int));
      err = hipMalloc((void **) &dev_bgij[i], N * N * sizeof(Complex));
      if (err != hipSuccess) {
        printf("failed to allocate dev_bgij[%d], size=%d, err=%d\n",
               i, N * N * sizeof(Complex), err);
        exit(1);
      }



#ifdef BUILDKKRMATRIX_GPU
      // hipMalloc((void**)&dev_bgij[i],4*kkrsz_max*kkrsz_max*numLIZ*numLIZ*sizeof(Complex));
      hipMalloc((void**)&dev_tmat_n[i],nspin*nspin*kkrsz_max*kkrsz_max*numLIZ*sizeof(Complex));
#endif

      hipMalloc((void **) &dev_tau[i], nspin * N * kkrsz_max * sizeof(Complex));
      hipMalloc((void **) &dev_tau00[i], nspin * nspin * kkrsz_max * kkrsz_max * sizeof(Complex));
#ifndef ARCH_IBM
      hipMalloc((void **) &dev_t[i], nspin  * N * kkrsz_max * sizeof(Complex));
#endif
      hipMalloc((void **) &dev_t0[i], nspin * nspin * kkrsz_max * kkrsz_max * sizeof(Complex));
      hipStreamCreate(&stream[i][0]);
      hipStreamCreate(&stream[i][1]);
      hipEventCreateWithFlags(&event[i], hipEventDisableTiming);
      cublasCheckError(hipblasCreate(&cublas_h[i]));
      cusolverCheckError(hipsolverDnCreate(&cusolverDnHandle[i]));
      int lWork;
      cusolverCheckError(hipsolverDnZgetrf_bufferSize(cusolverDnHandle[i], N, N,
                                                     (hipDoubleComplex *) dev_m[i], N, &lWork));
      dev_workBytes[i] = 0;
#ifndef ARCH_IBM
      hipsolverDnZZgesv_bufferSize(cusolverDnHandle[i], N, nspin * kkrsz_max,
                                  (hipDoubleComplex *) dev_m[i], N, dev_ipvt[i], (hipDoubleComplex *) dev_tau[i], N,
                                  (hipDoubleComplex *) dev_tau[i], N,
                                  dev_work[i], &dev_workBytes[i]);
#endif
      dev_workBytes[i] = std::max(dev_workBytes[i],
                                  lWork * sizeof(hipDoubleComplex));
#ifdef USE_XGETRF
      {
#if CUDA_VERSION < 11010
      printf("Error: Xgetrf requires CUDA 11.1+\n");
      exit(0);
#endif
      cusolverCheckError(hipsolverDnCreateParams(&cusolverDnParams[i]));
      cusolverCheckError(hipsolverDnSetAdvOptions(cusolverDnParams[i], HIPSOLVERDN_GETRF, CUSOLVER_ALG_2));
      hipMalloc((void**)&dev_ipvt64[i],N*sizeof(int64_t));
      size_t llWork;
      cusolverCheckError(hipsolverDnXgetrf_bufferSize(cusolverDnHandle[i],
                                                     cusolverDnParams[i],
                                                     (int64_t)N,
                                                     (int64_t)N,
                                                     HIP_C_64F,
                                                     (hipDoubleComplex *)dev_m[i],
                                                     (int64_t)N,
                                                     HIP_C_64F,
                                                     &llWork,
                                                     &host_workBytes[i]));
      dev_workBytes[i] = std::max(dev_workBytes[i], llWork);
      host_work[i] = malloc(host_workBytes[i]);
      }
#endif
#ifdef USE_IRSXGESV
      {
#if CUDA_VERSION < 10020
      printf("Error: IRSXgesv requires CUDA 10.2+\n");
      exit(1);
#endif
      cusolverCheckError(cusolverDnIRSParamsCreate(&cusolverDnIRSParams[i]));
      cusolverCheckError(cusolverDnIRSParamsSetRefinementSolver(cusolverDnIRSParams[i],
                                                                CUSOLVER_IRS_REFINE_CLASSICAL));
      cusolverCheckError(cusolverDnIRSParamsSetSolverPrecisions(cusolverDnIRSParams[i],
                                                                CUSOLVER_C_64F,
                                                                CUSOLVER_C_16F));
      cusolverCheckError(cusolverDnIRSInfosCreate(&cusolverDnIRSInfo[i]));
      size_t llWork;
      cusolverCheckError(cusolverDnIRSXgesv_bufferSize(cusolverDnHandle[i],
                                                       cusolverDnIRSParams[i],
                                                       N,
                                                       2*kkrsz_max,
                                                       &llWork));
      dev_workBytes[i] = std::max(dev_workBytes[i], llWork);
      hipMalloc((void**)&dev_X[i], nspin*nspin*N*kkrsz_max*sizeof(Complex));
      }
#endif
	hipMalloc((void**)&dev_work[i], dev_workBytes[i]);
        // printf("  dev_m[%d]=%zx\n",i,dev_m[i]);
      }
      deviceCheckError();
      initialized=true;
    }
    return 0;
  }

  void DeviceStorage::free()
  {
    if(initialized) {
   //     printf("*************************************MEMORY IS BEING FREED\n");
      // for(int i=0;i<omp_get_max_threads();i++)
      for(int i=0; i<nThreads; i++)
      {
        hipFree(dev_m[i]);
        hipFree(dev_ipvt[i]);
        hipFree(dev_info[i]);
        hipFree(dev_bgij[i]);
#ifdef BUILDKKRMATRIX_GPU
        hipFree(dev_tmat_n[i]);
#endif
	hipFree(dev_work[i]);
        hipFree(dev_t0[i]);
        hipStreamDestroy(stream[i][0]);
        hipStreamDestroy(stream[i][1]);
        hipEventDestroy(event[i]);
        hipblasDestroy(cublas_h[i]);
#ifdef USE_XGETRF
        ::free(host_work[i]);
        hipFree(dev_ipvt64[i]);
        hipsolverDnDestroyParams(cusolverDnParams[i]);
#endif
#ifdef USE_IRSXGESV
        cusolverDnIRSInfosDestroy(cusolverDnIRSInfo[i]);
        cusolverDnIRSParamsDestroy(cusolverDnIRSParams[i]);
        hipFree(dev_X[i]);
#endif
        hipsolverDnDestroy(cusolverDnHandle[i]);
      }
      // dev_tmat_store.clear();
      hipFree(devTmatStore);
      deviceCheckError();
      initialized=false;
    }
  }

/*
  static Complex* getDevM() { return dev_m[omp_get_thread_num()]; } 
  static Complex* getDevBGij() { if(!initialized) {printf("DeviceStorage not initialized\n"); exit(1);}
                                 return dev_bgij[omp_get_thread_num()]; } 
  static Complex* getDevTmatN() { return dev_tmat_n[omp_get_thread_num()]; } 
  static Complex* getDevTau() { return dev_tau[omp_get_thread_num()]; }
  static Complex* getDevTau00() { return dev_tau00[omp_get_thread_num()]; }
  static int* getDevIpvt() { return dev_ipvt[omp_get_thread_num()]; } 
  static hipStream_t getStream(int i) { return stream[omp_get_thread_num()][i]; }
  static hipEvent_t getEvent() { return event[omp_get_thread_num()]; }
  static hipblasHandle_t getCublasHandle() { return hipblas.h[omp_get_thread_num()]; }
  static hipsolverHandle_t getCusolverDnHandle() { return cusolverDnHandle[omp_get_thread_num()]; }
  static size_t getDevWorkBytes() { return dev_workBytes[omp_get_thread_num()]; }
  static void *getDevWork() {  return dev_work[omp_get_thread_num()]; }
  static DeviceMatrix<Complex>* getDevTmatStore() { return &dev_tmat_store; }
};
*/

int DeviceStorage::copyTmatStoreToDevice(Matrix<Complex> &tmatStore,
    int blkSize)
{
  if((tmatStoreSize > 0) && (tmatStoreSize < tmatStore.size()))
  {
    hipFree(devTmatStore);
    tmatStoreSize = 0;
  }
  if(tmatStoreSize == 0)
  {
    hipMalloc(&devTmatStore, tmatStore.size()*sizeof(Complex));
    tmatStoreSize = tmatStore.size();
  }
  hipMemcpy(devTmatStore, &tmatStore(0,0),
    tmatStore.size()*sizeof(Complex), hipMemcpyHostToDevice);
  blkSizeTmatStore = blkSize;
  tmatStoreLDim = tmatStore.l_dim();

  return 0;
}

bool DeviceStorage::initialized = false;
Complex *DeviceStorage::dev_m[MAX_THREADS], *DeviceStorage::dev_bgij[MAX_THREADS], *DeviceStorage::dev_tmat_n[MAX_THREADS];
Complex *DeviceStorage::dev_tau[MAX_THREADS], *DeviceStorage::dev_tau00[MAX_THREADS];
Complex *DeviceStorage::dev_t0[MAX_THREADS];
Complex *DeviceStorage::dev_t[MAX_THREADS];
void *DeviceStorage::dev_work[MAX_THREADS];
size_t DeviceStorage::dev_workBytes[MAX_THREADS];
int *DeviceStorage::dev_ipvt[MAX_THREADS];
int *DeviceStorage::dev_info[MAX_THREADS];
hipblasHandle_t DeviceStorage::hipblas.h[MAX_THREADS];
hipsolverHandle_t DeviceStorage::cusolverDnHandle[MAX_THREADS];
#ifdef USE_XGETRF
hipsolverDnParams_t DeviceStorage::cusolverDnParams[MAX_THREADS];
int64_t* DeviceStorage::dev_ipvt64[MAX_THREADS];
void* DeviceStorage::host_work[MAX_THREADS];
size_t DeviceStorage::host_workBytes[MAX_THREADS];
#endif
#ifdef USE_IRSXGESV
cusolverDnIRSParams_t DeviceStorage::cusolverDnIRSParams[MAX_THREADS];
cusolverDnIRSInfos_t DeviceStorage::cusolverDnIRSInfo[MAX_THREADS];
Complex *DeviceStorage::dev_X[MAX_THREADS];
#endif
hipEvent_t DeviceStorage::event[MAX_THREADS];
hipStream_t DeviceStorage::stream[MAX_THREADS][2];
// DeviceMatrix<Complex> DeviceStorage::dev_tmat_store;
Complex *DeviceStorage::devTmatStore;
size_t DeviceStorage::tmatStoreSize = 0;
int DeviceStorage::blkSizeTmatStore = 0;
int DeviceStorage::tmatStoreLDim = 0;
int DeviceStorage::nThreads=1;
bool initialized = false;

std::vector<DeviceAtom> deviceAtoms;

// Device Atom
int DeviceAtom::allocate(int _lmax, int _nspin, int _numLIZ)
{
  if(allocated) free();
  allocated = true;
  numLIZ = _numLIZ;
  hipMalloc((void**)&LIZPos,numLIZ*3*sizeof(Real));
  hipMalloc((void**)&LIZlmax,numLIZ*sizeof(int));
  hipMalloc((void**)&LIZStoreIdx,numLIZ*sizeof(int));

  return 0;
}

void DeviceAtom::free()
{
  if(allocated)
  {
    hipFree(LIZPos);
    hipFree(LIZlmax);
    hipFree(LIZStoreIdx);
  }
  allocated = false;
}

void DeviceAtom::copyFromAtom(AtomData &atom)
{
  if(!allocated)
  {
    allocate(atom.lmax, atom.nspin, atom.numLIZ);
  }
  hipMemcpy(LIZPos, &atom.LIZPos(0,0), atom.numLIZ*3*sizeof(Real), hipMemcpyHostToDevice);
  hipMemcpy(LIZlmax, &atom.LIZlmax[0], atom.numLIZ*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(LIZStoreIdx, &atom.LIZStoreIdx[0], atom.numLIZ*sizeof(int), hipMemcpyHostToDevice);
}

int *DeviceConstants::lofk;
int *DeviceConstants::mofk;
hipDoubleComplex *DeviceConstants::ilp1;
// DeviceMatrix<Complex> illp(ndlj, ndlj);
hipDoubleComplex* DeviceConstants::illp;
int DeviceConstants::ndlj_illp;
// DeviceArray3d<Real> cgnt(lmax+1,ndlj,ndlj);
Real* DeviceConstants::cgnt;
int DeviceConstants::ndlj_cgnt, DeviceConstants::lmaxp1_cgnt;

int DeviceConstants::allocate(AngularMomentumIndices &am, GauntCoeficients &c, IFactors &ifactors)
{
  ndlj_illp = ifactors.illp.l_dim();
  lmaxp1_cgnt = c.cgnt.l_dim1();
  ndlj_cgnt = c.cgnt.l_dim2();

  hipMalloc((void**)&lofk, am.lofk.size()*sizeof(int));
  hipMalloc((void**)&mofk, am.mofk.size()*sizeof(int));
  hipMalloc((void**)&ilp1, ifactors.ilp1.size()*sizeof(hipDoubleComplex));
  hipMalloc((void**)&illp, ifactors.illp.size()*sizeof(hipDoubleComplex));
  hipMalloc((void**)&cgnt, c.cgnt.size()*sizeof(double));

  hipMemcpy(lofk, &am.lofk[0], am.lofk.size()*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(mofk, &am.mofk[0], am.mofk.size()*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(ilp1, &ifactors.ilp1[0], ifactors.ilp1.size()*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
  hipMemcpy(illp, &ifactors.illp[0], ifactors.illp.size()*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
  hipMemcpy(cgnt, &c.cgnt[0], c.cgnt.size()*sizeof(double), hipMemcpyHostToDevice);

  return 0;
}

void DeviceConstants::free()
{
  hipFree(lofk);
  hipFree(mofk);
  hipFree(ilp1);
  hipFree(illp);
  hipFree(cgnt);
}


/****************Fortran Interfaces*********************/
extern "C"
Complex* get_dev_m_() {
  return DeviceStorage::getDevM();
}

extern "C"
Complex* get_dev_bgij_() {
  return DeviceStorage::getDevBGij();
}

extern "C"
Complex* get_dev_tmat_n_() {
  return DeviceStorage::getDevTmatN();
}

extern "C"
int* get_dev_ipvt_() {
  return DeviceStorage::getDevIpvt();
}

extern "C"
hipStream_t get_stream_(const int &id) {
  return DeviceStorage::getStream(id);
}

extern "C"
hipblasHandle_t get_cublas_handle_() {
  return DeviceStorage::getCublasHandle();
}

//allocate a thread specific event
extern "C"
hipEvent_t get_cuda_event_() {
  return DeviceStorage::getEvent();
}
/********************************************************/

// DeviceMatrix<Complex>* get_dev_tmat_store() {
//   return DeviceStorage::getDevTmatStore();
// }

void *allocateDStore(void)
{
  return static_cast<void *>(new DeviceStorage);
}

void freeDStore(void * d_store)
{
  static_cast<DeviceStorage*>(d_store)->free();
  delete static_cast<DeviceStorage*>(d_store);
}

int initDStore(void * d_store,int kkrsz_max, int nspin, int numLIZ, int nthreads)
{
  return (*static_cast<DeviceStorage*>(d_store)).allocate(kkrsz_max,nspin,numLIZ,nthreads);
}

// void copyTmatStoreToDevice(LocalTypeInfo &local) {
//  DeviceMatrix<Complex> &d_tmat_store=*get_dev_tmat_store();
//  d_tmat_store.copy_async(local.tmatStore,0);
// }
